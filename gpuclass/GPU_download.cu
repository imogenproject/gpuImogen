#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"
// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "nvToolsExt.h"

#include "cudaCommon.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	// At least 2 arguments expected
	// Input and result
	if((nlhs != 1) || ( (nrhs != 1) && (nrhs != 2) ) ) {
		mexErrMsgTxt("Form: host_array = GPU_download(GPU array, 'dump')");
	}

#ifdef USE_NVTX
	nvtxRangePush("Entering GPU_download");
#endif
	
	CHECK_CUDA_ERROR("entering GPU_download");
	
	MGArray m;
	
	int worked = MGA_accessMatlabArrays(prhs, 0, 0, &m);
	if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) {
		mexErrMsgTxt("GPU_download was not able to access GPU array.\n");
		return;
	}
	
	int singlePart;
	int nd = 3;
	mwSize odims[3];

	if(nrhs > 1) {
		singlePart = (int)*mxGetPr(prhs[1]); // only one partition
	} else {
		singlePart = -1;
	}

	if(singlePart >= 0) {
		int sub[6];
		calcPartitionExtent(&m, singlePart, &sub[0]);
		nd = 3;
		odims[0] = sub[3];
		odims[1] = sub[4];
		odims[2] = sub[5];
	} else {
		if(m.dim[2] == 1) {
			nd = 2;
			if(m.dim[1] == 1) {
				nd = 1;
			}
		}
		odims[0] = m.dim[0];
		odims[1] = m.dim[1];
		odims[2] = m.dim[2];
	}
	
	// Create output ueric array
	plhs[0] = mxCreateNumericArray(nd, odims, mxDOUBLE_CLASS, mxREAL);
	
	double *result = mxGetPr(plhs[0]);
	
	worked = MGA_downloadArrayToCPU(&m, &result, -1);

#ifdef USE_NVTX
	nvtxRangePop();
#endif

	if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) {
		mexErrMsgTxt("GPU_download accessed but couldn't download GPU array.");
		return;
	}
	return;
}
