#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "nvToolsExt.h"

#include "cudaCommon.h"
#include "cudaSoundspeed.h"

/* THIS FUNCTION:

   When not passed a magnetic field, calculates the local adiabatic sound speed of the fluid at 
   all points,

   c_s^2 = gamma*P/rho

   where c_s is the adiabatic sound speed, gamma is the adiabatic index (1 < gamma <= 5/3), P
   is the thermal pressure (gamma-1)*(Etotal - rho v^2/2) and rho is the matter density.

   When passed a magnetic field, calculates the maximal (field-aligned) magnetosonic velocity,
   C_fast^2 = C_s^2 + C_a^2,

   where C_s is the thermal sound speed above (Except subtracting magnetic energy density from
   the total energy as well) and C_a is the Alfven speed,

   C_a^2 = (B^2)/rho.
*/

__global__ void cukern_Soundspeed_mhd(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, int n);
__global__ void cukern_Soundspeed_hd(double *rho, double *E, double *px, double *py, double *pz, double *dout, int n);

#define BLOCKDIM 256
#define GRIDDIM  64

// FIXME: Not quite clear why this is 6 elements and not 2...
__device__ __constant__ double pressParams[6];
#define MHD_CS_B pressParams[0]
#define GG1 pressParams[1]

#ifdef STANDALONE_MEX_FUNCTION
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
	// Determine appropriate number of arguments for RHS
	if( (nlhs != 1) || ( (nrhs != 9) && (nrhs != 6) ))
		mexErrMsgTxt("calling form for cudaSoundspeed is c_s = cudaSoundspeed(mass, ener, momx, momy, momz, [bx, by, bz,] gamma);");

#ifdef USE_NVTX
        nvtxRangePush("Entering cudasoundspeed from mex");
#endif
	CHECK_CUDA_ERROR("entering cudaSoundspeed");

	// Select the appropriate kernel to invoke
	int pureHydro = (nrhs == 6);

	double gamma;
	MGArray fluid[5];
	MGArray mag[3];
	int worked;

	if(pureHydro == 1) {
		gamma    = *mxGetPr(prhs[5]);
		worked = MGA_accessMatlabArrays(prhs, 0, 4, &fluid[0]);
		if(worked != SUCCESSFUL) { DROP_MEX_ERROR("cudaSoundspeed.cu@67 failed to access arrays!"); }
		MGArray *dest = MGA_createReturnedArrays(plhs, 1, &fluid[0]);

		worked = calculateSoundspeed(&fluid[0], NULL, dest, gamma);
		if(worked != SUCCESSFUL) { DROP_MEX_ERROR("cudaSoundspeed.cu@71 dumped!"); }
		free(dest);
	} else {
		gamma = *mxGetPr(prhs[8]);
		worked = MGA_accessMatlabArrays(prhs, 0, 4, &fluid[0]);
		if(worked != SUCCESSFUL) { DROP_MEX_ERROR("cudaSoundspeed.cu@77 failed to access arrays!"); }
		worked = MGA_accessMatlabArrays(prhs, 5, 7, &mag[0]);
		if(worked != SUCCESSFUL) { DROP_MEX_ERROR("cudaSoundspeed.cu@79 failed to access arrays!"); }
		MGArray *dest = MGA_createReturnedArrays(plhs, 1, &fluid[0]);

		worked = calculateSoundspeed(&fluid[0], &mag[0], dest, gamma);
		if(worked != SUCCESSFUL) { DROP_MEX_ERROR("cudaSoundspeed.cu@82 dumped!"); }
		free(dest);
	}

	#ifdef SYNCMEX
		MGA_sledgehammerSequentialize(&fluid[0]);
	#endif

#ifdef USE_NVTX
        nvtxRangePop();
#endif

	return;
}
#endif

int calculateSoundspeed(MGArray *fluid, MGArray *mag, MGArray *cs, double gamma)
{
	double gg1 = gamma*(gamma-1);
	double hostParams[6];

	hostParams[0] = ALFVEN_CSQ_FACTOR - .5*gg1;
	hostParams[1] = gg1;

	dim3 blocksize; blocksize.x = BLOCKDIM; blocksize.y = blocksize.z = 1;
	dim3 gridsize;  gridsize.x = GRIDDIM;   gridsize.y = gridsize.z = 1;

	int pureHydro = (mag == NULL);

	int status = SUCCESSFUL;

	int i, j;
	int sub[6];
	double *srcs[8];
	for(i = 0; i < fluid[0].nGPUs; i++) {
		calcPartitionExtent(&fluid[0], i, &sub[0]);
		hipSetDevice(fluid[0].deviceID[i]);
		status = CHECK_CUDA_ERROR("Setting device.");
		if(status != SUCCESSFUL) { break; }

		hipMemcpyToSymbol((const void *)pressParams, &hostParams[0], 6*sizeof(double), 0, hipMemcpyHostToDevice);
		status = CHECK_CUDA_ERROR("cudaSoundspeed memcpy to constants.");
		if(status != SUCCESSFUL) { break; }

		for(j = 0; j < 5; j++) { srcs[j] = fluid[j].devicePtr[i]; }

		if(pureHydro == 1) {
			cukern_Soundspeed_hd<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], cs->devicePtr[i], fluid[0].partNumel[i]);
			status = CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, fluid, i, "cuda hydro soundspeed");
		} else {
			for(j = 0; j < 3; j++) { srcs[j+5] = mag[j].devicePtr[i]; }

			cukern_Soundspeed_mhd<<<gridsize, blocksize>>>(srcs[0], srcs[1], srcs[2], srcs[3], srcs[4], srcs[5], srcs[6], srcs[7], cs->devicePtr[i], fluid[0].partNumel[i]);
			status = CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, fluid, i, "cuda mhd soundspeed");
		}
		if(status != SUCCESSFUL) { break; }
	}

	return status;
}

// THIS KERNEL CALCULATES SOUNDSPEED IN THE MHD CASE, TAKEN AS THE FAST MA SPEED
// We increase the Alfven contribution to stabilize the code
__global__ void cukern_Soundspeed_mhd(double *rho, double *E, double *px, double *py, double *pz, double *bx, double *by, double *bz, double *dout, int n)
{

	int x = threadIdx.x + blockIdx.x * BLOCKDIM;
	int dx = blockDim.x * gridDim.x;
	double csq, T, Bsq;
	double invrho;

	while(x < n) {
		invrho = 1.0 / rho[x];
		T = .5*(px[x]*px[x] + py[x]*py[x] + pz[x]*pz[x])*invrho;
		Bsq = bx[x]*bx[x] + by[x]*by[x] + bz[x]*bz[x];

		// MHD_CS_B is (alfven constant A) - .5(gamma)(gamma-1), where A is physically 1
		// but may be increased beyond 1 to stabilize simulations where low-beta conditions occur
		csq = (GG1*(E[x] - T) + MHD_CS_B * Bsq ) * invrho ;
		if(csq < 0.0) csq = 0.0;
		dout[x] = sqrt(csq);
		x += dx;
	}

}

// THIS KERNEL CALCULATES SOUNDSPEED IN THE HYDRODYNAMIC CASE
__global__ void cukern_Soundspeed_hd(double *rho, double *E, double *px, double *py, double *pz, double *dout, int n)
{
	int x = threadIdx.x + blockIdx.x * BLOCKDIM;
	int dx = blockDim.x * gridDim.x;
	double csq, rhoinv;

	while(x < n) {
		rhoinv = 1/rho[x];
		csq = GG1*(E[x] - .5*(px[x]*px[x] + py[x]*py[x] + pz[x]*pz[x])*rhoinv)*rhoinv;
		// Imogen's energy flux is unfortunately not positivity preserving
		if(csq < 0.0) csq = 0.0;
		dout[x] = sqrt(csq);
		x += dx;
	}

}


