#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "hip/driver_types.h"

#include "cudaCommon.h"
#include "mpi_common.h"

/* THIS FILE, WHEN COMPILED, RESPONDS TO THE FOLLOWING -D FLAG:
 * -DALLOCFREE_DEBUG
 *     Causes every call to MGA_allocArrays and MGA_delete to emit a message to stdout;
 *     A poor man's valgrind.
 */

#define SYNCBLOCK 16

__global__ void cudaMGHaloSyncX_p2p(double *L, double *R, int nxL, int nxR, int ny, int nz, int h);
__global__ void cudaMGHaloSyncY_p2p(double *L, double *R, int nx, int nyL, int nyR, int nz, int h);

template<int lr_rw>
__global__ void cudaMGA_haloXrw(double *phi, double *linear, int nx, int ny, int nz, int h);

template<int lr_rw>
__global__ void cudaMGA_haloYrw(double *phi, double *linear, int nx, int ny, int nz, int h);

/* Given an mxArray* that points to a GPU tag (specifically the uint64_t array, not the more
 * general types tolerated by the higher-level functions), checks whether it can pass
 * muster as a MGArray (i.e. one packed by serializeMGArrayToTag).
 */
bool sanityCheckTag(const mxArray *tag)
{
	int64_t *x = (int64_t *)mxGetData(tag);

	int tagsize = mxGetNumberOfElements(tag);

	// This cannot possibly be valid
	if(tagsize < GPU_TAG_LENGTH) {
		printf("Tag length is %i < min possible valid length of %i. Dumping.\n", tagsize, GPU_TAG_LENGTH);
		return false;
	}

	int nx = x[GPU_TAG_DIM0];
	int ny = x[GPU_TAG_DIM1];
	int nz = x[GPU_TAG_DIM2];

	// Null array OK
	if((nx == 0) && (ny == 0) && (nz == 0) && (tagsize == GPU_TAG_LENGTH)) return true;

	if((nx < 0) || (ny < 0) || (nz < 0)) {
		printf("One or more indices was of negative size. Dumping.\n");
		return false;
	}

	int halo         = x[GPU_TAG_HALO];
	int partitionDir = x[GPU_TAG_PARTDIR];
	int nDevs        = x[GPU_TAG_NGPUS];

	int permtag      = x[GPU_TAG_DIMPERMUTATION];

	int circlebits   = x[GPU_TAG_CIRCULARBITS];

	int vecpart      = x[GPU_TAG_VECTOR_COMPONENT];

	// Some basic does-this-make-sense
	if(nDevs < 1) {
		printf((const char *)"Tag indicates less than one GPU in use.\n");
		return false;
	}
	if(nDevs > MAX_GPUS_USED) {
		printf((const char *)"Tag indicates %i GPUs in use, current config only supports %i.\n", nDevs, MAX_GPUS_USED);
		return false;
	}
	if(halo < 0) { // not reasonable.
		printf((const char *)"Tag halo value is %i < 0 which is absurd. Dumping.\n", halo);
		return false;
	}

	if((permtag < 1) || (permtag > 6)) {
		if(permtag == 0) {
			// meh
		} else {
			printf((const char *)"Permutation tag is %i: Valid values are 1 (XYZ), 2 (XZY), 3 (YXZ), 4 (YZX), 5 (ZXY), 6 (ZYX)\n", permtag);
			return false;
		}
	}

	if((circlebits < 0) || (circlebits > 63)) {
		printf((const char *)"halo sharing bits have value %i, valid range is 0-63!\n", circlebits);
		return false;

	}

	if((vecpart < 0) || (vecpart > 3)) {
		printf((const char *)"vector component has value %i, must be 0 (scalar) or 1/2/3 (x/y/z)!\n", vecpart);
		return false;
	}

	if((partitionDir < 1) || (partitionDir > 3)) {
		printf((const char *)"Indicated partition direction of %i is not 1, 2, or 3.\n", partitionDir);
		return false;
	}

	// Require there be enough additional elements to hold the physical device pointers & cuda device IDs
	int requisiteNumel = GPU_TAG_LENGTH + 2*nDevs;
	if(tagsize != requisiteNumel) {
		printf((const char *)"Tag length is %i: Must be %i base + 2*nDevs = %i\n", tagsize, GPU_TAG_LENGTH, requisiteNumel);
		return false;
	}

	int j;
	x += GPU_TAG_LENGTH;
	// CUDA device #s are nonnegative, and it is nonsensical that there would be over 16 of them.
	for(j = 0; j < nDevs; j++) {
		if((x[2*j] < 0) || (x[2*j] >= MAX_GPUS_USED)) {
			printf((const char *)"Going through .deviceID: Found %i < 0 or > %i is impossible. Dumping.\n", (int)x[2*j], MAX_GPUS_USED);
			return false;
		}
	}

	return true;
}

/* Write [x0 y0 z0 nx ny nz] to the first 6 elements of sub for partition P of the MGArray
 * pointed to by m.
 * DO NOT use ANY other function to compute MGA extents!!!
 */
void calcPartitionExtent(MGArray *m, int P, int *sub)
{
	if(P >= m->nGPUs) {
		char bugstring[256];
		sprintf(bugstring, (const char *)"Fatal: Requested partition %i but only %i GPUs in use.", P, m->nGPUs);
		mexErrMsgTxt(bugstring);
	}

	int direct = m->partitionDir - 1; // zero-indexed direction

	int i;
	// We get the whole array transverse to the partition direction
	for(i = 0; i < 3; i++) {
		if(i == direct) continue;
		sub[i  ] = 0;
		sub[3+i] = m->dim[i];
	}

	sub += direct;
	int alpha = m->dim[direct] / m->nGPUs;

	// "raw" offset of P*alpha, extent of alpha
	sub[0] = P*alpha;
	sub[3] = alpha;

	// Rightmost partition takes up any remainder slack
	if(P == (m->nGPUs-1)) sub[3] = m->dim[direct] - P*alpha;

	// MultiGPU operation requires halos on both sides of the partitions for FDing operations
	if(m->nGPUs > 1) {
		if((m->addExteriorHalo != 0) || (P > 0)) {
			sub[0] -= m->haloSize; sub[3] += m->haloSize;
		}
		if((m->addExteriorHalo != 0) || (P < (m->nGPUs-1))) {
			sub[3] += m->haloSize;
		}
	}

}

/* Given an address on a partition addr and the results of the partition it came from stored in
int sub[6], stores the address in the partition in *subAddr and the global address as will be seen
in Matlab in *globAddr */
void decodePartitionAddress(long addr, int *sub, int3 *subAddr, int3 *globAddr)
{
long partnxy = sub[3]*sub[4];

subAddr->z = addr / partnxy;

addr -= subAddr->z * partnxy;

subAddr->y = addr / sub[3];

subAddr->x = addr - sub[3] * subAddr->y;

globAddr->x = subAddr->x + sub[0];
globAddr->y = subAddr->y + sub[1];
globAddr->z = subAddr->z + sub[2];

}

/* Given an mxArray *X, it searches "all the places you'd expect" any function in Imogen to have
 * stored the uint64_t pointer to the tag itself. Specifically, if X is a:
 *   uint64_t class: returns mxGetData(X)
 *   GPU_Type class: returns mxGetData(mxGetProperty(X, 0, "GPU_MemPtr"));
 *   FluidArray    : returns mxGetData(mxGetProperty(X, 0, "gputag"));
 */
int getGPUTypeTag(const mxArray *gputype, int64_t **tagPointer)
{
	return getGPUTypeTagIndexed(gputype, tagPointer, 0);
}

/* Behaves as getGPUTypeTag, but can fetch outside of index zero. */
int getGPUTypeTagIndexed(const mxArray *gputype, int64_t **tagPointer, int mxarrayIndex)
{
	static int64_t locptr[GPU_TAG_MAXLEN];

	if(tagPointer == NULL) {
		PRINT_FAULT_HEADER;
		printf("input tag pointer was null!\n");
		PRINT_FAULT_FOOTER;
		return ERROR_NULL_POINTER;
	}
	tagPointer[0] = NULL;

	mxClassID dtype = mxGetClassID(gputype);

	/* Handle gpu tags straight off */
	if(dtype == mxINT64_CLASS) {
		bool sanity = sanityCheckTag(gputype);
		if(sanity == false) {
			PRINT_FAULT_HEADER;
			printf((const char *)"Failure to access GPU tag: Sanity check failed.\n");
			PRINT_FAULT_FOOTER;
			return ERROR_GET_GPUTAG_FAILED;
		}
		int64_t *q = (int64_t *)mxGetData(gputype);
		int ctr;
		for(ctr = 0; ctr < mxGetNumberOfElements(gputype); ctr++) {
			locptr[ctr] = q[ctr];
		}
		tagPointer[0] = &locptr[0];
		return SUCCESSFUL;
	}

	mxArray *tag;
	const char *cname = mxGetClassName(gputype);

	/* If we were passed a GPU_Type, retreive the GPU_MemPtr element */
	if(strcmp(cname, (const char *)"GPU_Type") == 0) {
		tag = mxGetProperty(gputype, mxarrayIndex, (const char *)"GPU_MemPtr");
	} else { /* Assume it's an ImogenArray or descendant and retrieve the gputag property */
		tag = mxGetProperty(gputype, mxarrayIndex, (const char *)"gputag");
	}

	/* We have done all that duty required, there is no dishonor in surrendering */
	if(tag == NULL) {
		PRINT_FAULT_HEADER;
		printf((const char *)"getGPUTypeTag was called with something that is not a gpu tag, or GPU_Type class, or ImogenArray class\nArgument order wrong?\n");
		PRINT_FAULT_FOOTER;
		return ERROR_CRASH;
	}

	bool sanity = sanityCheckTag(tag);
	if(sanity == false) {
		PRINT_FAULT_HEADER;
		printf((const char *)"Failure to access GPU tag: Sanity check failed.\n");
		PRINT_FAULT_FOOTER;
		return ERROR_GET_GPUTAG_FAILED;
	}

	int64_t *q = (int64_t *)mxGetData(tag);
	int ctr;
	for(ctr = 0; ctr < mxGetNumberOfElements(tag); ctr++) {
		locptr[ctr] = q[ctr];
	}
	tagPointer[0] = &locptr[0];
	mxDestroyArray(tag);
	//tagPointer[0] = (int64_t *)mxGetData(tag);

	return SUCCESSFUL;
}

int getGPUTypeStreams(const mxArray *fluidarray, hipStream_t **streams, int *numel) {
	mxArray *streamptr  = mxGetProperty(fluidarray, 0, (const char *)"streamptr");

	if(streamptr != NULL) {
		*numel = (int)mxGetNumberOfElements(streamptr);
		streams[0] = (hipStream_t *)mxGetData(streamptr);
		return 0;
	} else {
		*numel = 0;
		return 0;
	}
}

// SERDES: Unpacks a uint64_t vector into an MGArray
int deserializeTagToMGArray(int64_t *tag, MGArray *mg)
{
	if(tag == NULL) {
			PRINT_FAULT_HEADER;
			printf((const char *)"input tag pointer was null!\n");
			PRINT_FAULT_FOOTER;
			return ERROR_NULL_POINTER;
		}
	int i;
	mg->numel = 1;

	mg->dim[0] = tag[GPU_TAG_DIM0];
	mg->numel *= mg->dim[0];
	mg->dim[1] = tag[GPU_TAG_DIM1];
	mg->numel *= mg->dim[1];
	mg->dim[2] = tag[GPU_TAG_DIM2];
	mg->numel *= mg->dim[2];
	mg->numSlabs = tag[GPU_TAG_DIMSLAB];

	mg->haloSize     = tag[GPU_TAG_HALO];
	mg->partitionDir = tag[GPU_TAG_PARTDIR];
	mg->nGPUs        = tag[GPU_TAG_NGPUS];

	mg->addExteriorHalo = tag[GPU_TAG_EXTERIORHALO];

	mg->permtag = tag[GPU_TAG_DIMPERMUTATION];
    MGA_permtagToNums(mg->permtag, &mg->currentPermutation[0]);

    mg->vectorComponent = tag[GPU_TAG_VECTOR_COMPONENT];

    mg->circularBoundaryBits = tag[GPU_TAG_CIRCULARBITS];

	int sub[6];

	tag += GPU_TAG_LENGTH;
	for(i = 0; i < mg->nGPUs; i++) {
		mg->deviceID[i]  = (int)tag[2*i];
		mg->devicePtr[i] = (double *)tag[2*i+1];
		// Many elementwise funcs only need numel, so avoid having to do this every time
		calcPartitionExtent(mg, i, sub);
		mg->partNumel[i] = sub[3]*sub[4]*sub[5];
		mg->slabPitch[i] = ROUNDUPTO(mg->partNumel[i]*sizeof(double), 256);
	}
	for(; i < MAX_GPUS_USED; i++) {
		mg->deviceID[i]  = -1;
		mg->devicePtr[i] = 0x0;
		mg->partNumel[i] = 0;
		mg->slabPitch[i] = 0;
	}

	return SUCCESSFUL;
}

// SERDES: Packs an MGArray into a uint64_t vector
void serializeMGArrayToTag(MGArray *mg, int64_t *tag)
{
	tag[GPU_TAG_DIM0]    = mg->dim[0];
	tag[GPU_TAG_DIM1]    = mg->dim[1];
	tag[GPU_TAG_DIM2]    = mg->dim[2];
	tag[GPU_TAG_DIMSLAB] = mg->numSlabs;
	tag[GPU_TAG_HALO]    = mg->haloSize;
	tag[GPU_TAG_PARTDIR] = mg->partitionDir;
	tag[GPU_TAG_NGPUS]   = mg->nGPUs;
	tag[GPU_TAG_EXTERIORHALO]    = mg->addExteriorHalo;
	tag[GPU_TAG_DIMPERMUTATION]  = mg->permtag;
	tag[GPU_TAG_CIRCULARBITS]    = mg->circularBoundaryBits;
	tag[GPU_TAG_VECTOR_COMPONENT]= mg->vectorComponent;

	int i;
	for(i = 0; i < mg->nGPUs; i++) {
		tag[GPU_TAG_LENGTH+2*i]   = (int64_t)mg->deviceID[i];
		tag[GPU_TAG_LENGTH+2*i+1] = (int64_t)mg->devicePtr[i];
	}

	return;
}

/* Converts the MGArray's .permtag element which uniquely specifies the current in-memory layout
 * into p[3] such that p[i] gives the 'physical' direction which lies in the i direction
 * in memory, for convenience.
 * */
void MGA_permtagToNums(int permtag, int *p)
{
	if(p == NULL) return;

	switch(permtag) {
	case 1: p[0] = 1; p[1] = 2; p[2] = 3; break;
	case 2: p[0] = 1; p[1] = 3; p[2] = 2; break;
	case 3: p[0] = 2; p[1] = 1; p[2] = 3; break;
	case 4: p[0] = 2; p[1] = 3; p[2] = 1; break;
	case 5: p[0] = 3; p[1] = 1; p[2] = 2; break;
	case 6: p[0] = 3; p[1] = 2; p[2] = 1; break;
	}

}

/* Reverse of permtagToNums: Routines which rotate memory and alter the currentPermutation[]
 * should use this to make sure that .permtag, which actually represents it, is updated too
 */
int MGA_numsToPermtag(int *nums)
{
	if(nums == NULL) return -1;

	switch(nums[0]) {
	case 1: { // x first
		if((nums[1] == 2) && (nums[2] == 3)) return 1; // XYZ
		if((nums[1] == 3) && (nums[2] == 2)) return 2; // XZY
	} break;
	case 2: { // y first
		if((nums[1] == 1) && (nums[2] == 3)) return 3; // YXZ
		if((nums[1] == 3) && (nums[2] == 1)) return 4; // YXZ
	} break;
	case 3: { // z first
		if((nums[1] == 1) && (nums[2] == 2)) return 5; // ZXY
		if((nums[1] == 2) && (nums[2] == 1)) return 6; // ZYX
	} break;
	}

return 0;
}

int MGA_dir2memdir(int *perm, int dir)
{
	if(perm == NULL) return -1;

	if(perm[0] == dir) return 1;
	if(perm[1] == dir) return 2;
	if(perm[2] == dir) return 3;

	return -1;
}

/* Facilitates access to MGArrays stored in Imogen's Matlab structures:
 * the mxArray pointers prhs[i] for i spanning idxFrom to idxTo inclusive
 * are decoded into mg[i - idxFrom]. Such that if the Matlab call is
 *    matlabFoo(1, 2, gpuA, gpuB, gpuC)
 * then foo(const mxArray *prhs[], ...) should use
 *    MGA_accessMatlabArrays(prhs, 2, 4, x)
 * with the result that x[0] = gpuA, x[1] = gpuB, x[2] = gpuC.
 */
int MGA_accessMatlabArrays(const mxArray *prhs[], int idxFrom, int idxTo, MGArray *mg)
{
	int i;
	int returnCode = SUCCESSFUL;
	prhs += idxFrom;

	int64_t *tag;

	for(i = 0; i < (idxTo + 1 - idxFrom); i++) {
		    if(prhs[i] == NULL) {
		    	PRINT_FAULT_HEADER;
		    	printf("Reading array #%i: prhs[i] was NULL (from=%i, to=%i)\n", i, idxFrom, idxTo);
		    	PRINT_FAULT_FOOTER;
		    	return ERROR_NULL_POINTER;
		    }
			returnCode = getGPUTypeTag(prhs[i], &tag);

			if(returnCode == SUCCESSFUL)
				returnCode = deserializeTagToMGArray(tag, &mg[i]);

			mg[i].matlabClassHandle = prhs[i]; // FIXME: This is a craptastic hack
			// I am too lazy to implement boundary condition data storage properly...
			mg[i].mlClassHandleIndex = 0;

			if(returnCode != SUCCESSFUL) break;
		}

	return CHECK_IMOGEN_ERROR(returnCode);
}

/* Facilitates access to vector GPU array arguments from Matlab. Such that if
 *   matlab>> x = [gpuA gpuB gpuC];
 *   matlab>> matlabFoo(1, x, stuff)
 * Then foo(const mxArray *prhs, ...) should use
 *   MGA_accessMatlabArrayVector(prhs[1], 0, 2, z)
 * with the result that z[0] = gpuA, z[1] = gpuB, z[2] = gpuC.
 */
int MGA_accessMatlabArrayVector(const mxArray *m, int idxFrom, int idxTo, MGArray *mg)
{

	int i;
	int returnCode = SUCCESSFUL;

	int64_t *tag;

	for(i = 0; i < (idxTo + 1 - idxFrom); i++) {
			returnCode = getGPUTypeTagIndexed(m, &tag, i);

			if(returnCode == SUCCESSFUL)
				returnCode = deserializeTagToMGArray(tag, &mg[i]);

			mg[i].matlabClassHandle = m; // FIXME: This is a craptastic hack
			// I am too lazy to implement boundary condition data storage properly...
			mg[i].mlClassHandleIndex = i;

			if(returnCode != SUCCESSFUL) break;
		}

	return CHECK_IMOGEN_ERROR(returnCode);
}

/* Given a pointer to a template array 'skeleton', returns a vector of N MGArrays whose
 * size and partitioning match that of skeleton. If skeleton is a slab referent, new arrays
 * are real arrays the size of one slab (numSlabs = 1).
 */
int MGA_allocArrays(MGArray **ret, int N, MGArray *skeleton)
{
	// Do some preliminaries,
	MGArray *m = (MGArray *)malloc(N*sizeof(MGArray));
	if(m == NULL) {
		PRINT_FAULT_HEADER;
		printf("Unable to malloc(N*sizeof(MGArray); N = %i", N);
		PRINT_FAULT_FOOTER;
		return ERROR_NOMEM;
	}

	int i, j, worked, sub[6];

	/* If we are passed a slab array (e.g. the second slab of a 5-slab set),
	 * allocate this array to be a single-slab array (i.e. assume that unless
	 * explicitly stated otherwise, "make new a new array like skeleton" means
	 * one slab element, not the whole thing.
	 */
	int nActualSlabs = skeleton->numSlabs;
	if(nActualSlabs <= 0) nActualSlabs = 1;

	// clone skeleton,
	for(i = 0; i < N; i++) {
		m[i]       = *skeleton;
		m[i].numSlabs = nActualSlabs;

		// but all "derived" qualities need to be reset
		m[i].numel = m[i].dim[0]*m[i].dim[1]*m[i].dim[2];

		// allocate new memory
		for(j = 0; j < skeleton->nGPUs; j++) {
			hipSetDevice(m[i].deviceID[j]);
			worked = CHECK_CUDA_ERROR("MGA_allocArrays: hipMalloc");
			if(worked != SUCCESSFUL) break;
			m[i].devicePtr[j] = 0x0;

			// Check this, because the user may have merely set .haloSize = PARTITION_CLONED
			calcPartitionExtent(m+i, j, sub);
			m[i].partNumel[j] = sub[3]*sub[4]*sub[5];
			m[i].slabPitch[j] = ROUNDUPTO(m[i].partNumel[j]*sizeof(double), 256);

			/* Differs if we have slabs... */
			int64_t num2alloc = m[i].partNumel[j] * sizeof(double);
			if(m[i].numSlabs > 1) num2alloc = m[i].slabPitch[j];

			hipMalloc((void **)&m[i].devicePtr[j], num2alloc);
			worked = CHECK_CUDA_ERROR("MGA_allocArrays: hipMalloc");
			if(worked != SUCCESSFUL) break;
		}
		if(worked != SUCCESSFUL) break;
	}

	if(worked != SUCCESSFUL) {
		printf("GPU array allocation process failed: device identifier %i, array %i/%i\n", j, i, N);
		PRINT_FAULT_FOOTER;
	}

#ifdef ALLOCFREE_DEBUG
printf((const char *)"============= MGA_allocArrays invoked\n");
printf((const char *)"Creating %i arrays\n", N);
printf((const char *)"Array ptr: %lx\n", (unsigned long)m);
for(i = 0; i < N; i++) {
	for(j = 0; j < m[i].nGPUs; j++) printf((const char *)"	Pointer %i: %lx\n", m[i].deviceID[j], (unsigned long)m[i].devicePtr[j]);
}
#endif

	*ret = m;
	return SUCCESSFUL;
}

/* Given a *skeleton to use as a template, allocates a new array into nu[0],
 * which is identical to *skeleton except that it has Nslabs slabs. The memory
 * has proper CUDA striding (slab N+1 starts an even multiple of 256B after slab N does)
 * and is not initialized in any way. */
int MGA_allocSlab(MGArray *skeleton, MGArray *nu, int Nslabs)
{
	int sub[6];
	// Do the allocate-and-copy dance since we don't have a cudaRealloc that I know of
	int i;
	int worked;

	*nu = *skeleton; // copy everything since nothing but the devicePtr[] & numSlabs will change

	for(i = 0; i < skeleton->nGPUs; i++) {
		hipSetDevice(skeleton->deviceID[i]);
		worked = CHECK_CUDA_ERROR("setdevice");
		if(worked != SUCCESSFUL) break;

		calcPartitionExtent(skeleton, i, &sub[0]);
		// number of bytes per slab
		int64_t slabsize = sub[3]*sub[4]*sub[5] * sizeof(double);
		// round up to make a pleasantly CUDA-aligned amount
		int64_t slabpitch = slabsize / 256;
		slabpitch += (256*slabpitch < slabsize); slabpitch *= 256;
		nu->slabPitch[i] = slabpitch; // just to be safe, this should be correct already

		hipMalloc((void **)&nu->devicePtr[i], slabpitch*Nslabs);
		worked = CHECK_CUDA_ERROR("malloc");
		if(worked != SUCCESSFUL) break;
	}
	nu->numSlabs = Nslabs;

#ifdef ALLOCFREE_DEBUG
printf((const char *)"======== MGA_allocSlab invoked\n");
printf((const char *)"Creating    %i slabs\n", Nslabs);
printf((const char *)"Template *: %lx\n", (unsigned long)skeleton);
printf((const char *)"Dest *:     %lx\n", (unsigned long)nu);
int j;
for(i = 0; i < Nslabs; i++) {
	for(j = 0; j < nu->nGPUs; j++) printf((const char *)"	Pointer %i: %lx\n", nu->deviceID[j], (unsigned long)(nu->devicePtr[j]+j*nu->slabPitch[j]/sizeof(double)));
}
#endif

	if(worked != SUCCESSFUL) {
		printf("\n	!!! MGA_allocSlab is now bailing with error: Program probably crashing hard !!!\n");
	}

	return CHECK_IMOGEN_ERROR(worked);
}

int MGA_duplicateArray(MGArray **dst, MGArray *src)
{
	int status = SUCCESSFUL;
	if((src == NULL) || (dst == NULL)) {
		PRINT_FAULT_HEADER;
		printf("Null src argument passed to MGA_duplicateArray. Crashing.\n");
		PRINT_FAULT_FOOTER;
		return ERROR_NULL_POINTER;
	}

	if(dst[0] == NULL) {
		status = MGA_allocArrays(&dst[0], 1, src);
		if(CHECK_IMOGEN_ERROR(status) != SUCCESSFUL) return status;
	}

	int i;
	for(i = 0; i < src->nGPUs; i++) {
		hipSetDevice(src->deviceID[i]);
		if(CHECK_CUDA_ERROR("Array duplicator:hipSetDevice") != SUCCESSFUL) {
					status = ERROR_CRASH;
					break;
				}

		hipError_t unhappy = hipMemcpy((void *)dst[0]->devicePtr[i], (void *)src->devicePtr[i], src->partNumel[i]*sizeof(double), hipMemcpyDeviceToDevice);
		if(CHECK_CUDA_ERROR("Array duplicator:hipMemcpy") != SUCCESSFUL) {
			status = ERROR_CRASH;
			break;
		}
	}
	return status;
}

/* A convenient wrapper for returning data to Matlab:
 * Creates arrays in the style of MGA_allocArrays, but serializes them into the
 * first N elements of plhs[] as well before returning the C vector pointer.
 */
MGArray *MGA_createReturnedArrays(mxArray *plhs[], int N, MGArray *skeleton)
{

	MGArray *m;
	int status = MGA_allocArrays(&m, N, skeleton);

	int i;

	mwSize dims[2]; dims[0] = GPU_TAG_LENGTH+2*skeleton->nGPUs; dims[1] = 1;
	int64_t *r;

	// create Matlab arrays holding serialized form,
	for(i = 0; i < N; i++) {
		plhs[i] = mxCreateNumericArray(2, dims, mxINT64_CLASS, mxREAL);
		r = (int64_t *)mxGetData(plhs[i]);
		serializeMGArrayToTag(m+i, r);
	}

	// send back the MGArray structs.
	return m;
}

/* Does exactly what it says: Accepts one MGArray pointer and
 * writes one serialized representation to plhs[0].
 */
void MGA_returnOneArray(mxArray *plhs[], MGArray *m)
{
	mwSize dims[2]; dims[0] = GPU_TAG_LENGTH+2*m->nGPUs; dims[1] = 1;
	int64_t *r;

	// create Matlab arrays holding serialized form,
	plhs[0] = mxCreateNumericArray(2, dims, mxINT64_CLASS, mxREAL);
	r = (int64_t *)mxGetData(plhs[0]);
	serializeMGArrayToTag(m, r);
}

/* Deallocates an MGArray by freeing all its devicePtr[] entries.
 * If it is passed a slab reference (numSlabs < 1), does nothing */
int MGA_delete(MGArray *victim)
{
	if(victim == NULL) {
		PRINT_FAULT_HEADER;
		printf("MGA_delete passed a null MGA to delete!\n");
		PRINT_FAULT_FOOTER;
		return ERROR_NULL_POINTER;
	}
	if(victim->numSlabs < 1) return SUCCESSFUL; // Ignore attempts to deallocate slab refs, this lets us pretend they're "normal"

	int returnCode = SUCCESSFUL;
	int j = 0;

#ifdef ALLOCFREE_DEBUG
printf((const char *)"MGA_delete invoked ==============\n");
printf((const char *)"Victim *: %lx\n", (unsigned long)victim);
for(j = 0; j < victim->nGPUs; j++) {
	printf((const char *)"	Device: %i, ptr %lx\n", victim->deviceID[j], (unsigned long)victim->devicePtr[j]);
}
fflush(stdout);
#endif

	for(j = 0; j<victim->nGPUs; j++){
		hipSetDevice(victim->deviceID[j]);
		returnCode = CHECK_CUDA_ERROR((const char *)"In MGA_delete, setting device");

		hipFree(victim->devicePtr[j]);
		if(returnCode == SUCCESSFUL) returnCode = CHECK_CUDA_ERROR((const char *)"In MGA_delete after hipFree");

		if(returnCode != SUCCESSFUL) break;
	}

	return CHECK_IMOGEN_ERROR(returnCode);
}

/* Some routines still run in the mold of "we were passed N arrays so expect N pointers"
 * and this makes it simple enough to do this.
 */
void pullMGAPointers( MGArray *m, int N, int i, double **dst)
{
	int x;
	for(x = 0; x < N; x++) { dst[x] = m[x].devicePtr[i]; }
}

int3 makeInt3(int x, int y, int z) {
	int3 a; a.x = x; a.y = y; a.z = z; return a; }
int3 makeInt3(int *b) {
	int3 a; a.x = b[0]; a.y = b[1]; a.z = b[2]; return a; }
dim3 makeDim3(unsigned int x, unsigned int y, unsigned int z) {
	dim3 a; a.x = x; a.y = y; a.z = z; return a; }
dim3 makeDim3(unsigned int *b) {
	dim3 a; a.x = b[0]; a.y = b[1]; a.z = b[2]; return a; }
dim3 makeDim3(int *b) {
	dim3 a; a.x = (unsigned int)b[0]; a.y = (unsigned int)b[1]; a.z = (unsigned int)b[2]; return a; }

/* This function should only be used for debugging race conditions
 * It loops over q->deviceID[] and calls hipDeviceSynchronize() for each. */
void MGA_sledgehammerSequentialize(MGArray *q)
{
	int i;
	for(i = 0; i < q->nGPUs; i++) {
		hipSetDevice(q->deviceID[i]);
		hipDeviceSynchronize();
	}
}


double cpu_reduceInitValue(MGAReductionOperator op)
{
	switch(op) {
	case MGA_OP_SUM:  return 0.0;
	case MGA_OP_PROD: return 1.0;
	case MGA_OP_MIN:  return 1e37;
	case MGA_OP_MAX:  return -1e37;
	}
	return NAN;
}

double cpu_reducePair(double A, double B, MGAReductionOperator op)
{
	switch(op) {
	case MGA_OP_SUM:  return A+B;
	case MGA_OP_PROD: return A*B;
	case MGA_OP_MIN:  return (A < B) ? A : B;
	case MGA_OP_MAX:  return (A > B) ? A : B;
	}
	return NAN;
}

// NVCC should optimize these to a single register load because they're called by a templated function only...
__device__ double cukern_reduceInitValue(MGAReductionOperator op)
{
	switch(op) {
	case MGA_OP_SUM:  return 0.0;
	case MGA_OP_PROD: return 1.0;
	case MGA_OP_MIN:  return 1e37;
	case MGA_OP_MAX:  return -1e37;
	}
	return NAN;
}

__device__ double cukern_reducePair(double A, double B, MGAReductionOperator op)
{
	switch(op) {
	case MGA_OP_SUM:  return A+B;
	case MGA_OP_PROD: return A*B;
	case MGA_OP_MIN:  return (A < B) ? A : B;
	case MGA_OP_MAX:  return (A > B) ? A : B;
	}
	return NAN;
}

template <MGAReductionOperator OPERATION>
__global__ void cukern_reduceScalar(double *phi, double *retvals, int n)
{
	unsigned int tix = threadIdx.x;
	int x = blockIdx.x * blockDim.x + tix;

	__shared__ double W[256];

	double Wmax = -1e37;
	W[tix] = -1e37;
	if(tix == 0) retvals[blockIdx.x] = Wmax; // As a safety measure incase we return below

	if(x >= n) return; // If we're fed a very small array, this will be easy

	// Threads step through memory with a stride of (total # of threads), finphig the max in this space
	while(x < n) {
	  if(phi[x] > Wmax) Wmax = phi[x];
	  x += blockDim.x * gridDim.x;
	  }
	W[tix] = Wmax;

	x = 128;
	while(x > 16) {
		if(tix >= x) return;
		__syncthreads();
		W[tix] = cukern_reducePair(W[tix],W[tix+x], OPERATION);
		x=x/2;
	}

	__syncthreads();

	// We have one halfwarp (16 threads) remaining
	// Assume that warps behave SIMD-synchronously
	W[tix] = cukern_reducePair(W[tix],W[tix+16], OPERATION); if(tix >= 8) return;
	W[tix] = cukern_reducePair(W[tix],W[tix+8], OPERATION); if(tix >= 4) return;
	W[tix] = cukern_reducePair(W[tix],W[tix+4], OPERATION); if(tix >= 2) return;
	W[tix] = cukern_reducePair(W[tix],W[tix+2], OPERATION); if(tix) return;

	retvals[blockIdx.x] = cukern_reducePair(W[0],W[1], OPERATION);
}

/* MGA_localReduceScalar uses the GPU to compute the reduction of the array described by
 * 'in' to a single value and writes that to scalar[0].
 */
int MGA_localReduceScalar(MGArray *in, double *scalar, MGAReductionOperator operate)
{
	int returnCode;
	dim3 blocksize, gridsize;
	blocksize.x = 256; blocksize.y = blocksize.z = 1;

	gridsize.x = 32; // 8K threads ought to keep the bugger busy
	gridsize.y = gridsize.z =1;

	// Allocate gridsize elements of pinned memory per GPU
	// Results will be conveniently waiting on the CPU for us when we're done
	double *blockValues[in->nGPUs];

	int i;
	for(i = 0; i < in->nGPUs; i++) {
		hipSetDevice(in->deviceID[i]);
		returnCode = CHECK_CUDA_ERROR((const char *)"calling hipSetDevice()");
		if(returnCode != SUCCESSFUL) break;

		hipHostMalloc(&blockValues[i], gridsize.x * sizeof(double));
		returnCode = CHECK_CUDA_ERROR((const char *)"hipHostMalloc");
		if(returnCode != SUCCESSFUL) break;
		switch(operate) {
				case MGA_OP_SUM:  cukern_reduceScalar<MGA_OP_SUM><<<gridsize, blocksize>>>(in->devicePtr[i], blockValues[i], in->partNumel[i]); break;
				case MGA_OP_PROD: cukern_reduceScalar<MGA_OP_PROD><<<gridsize, blocksize>>>(in->devicePtr[i], blockValues[i], in->partNumel[i]); break;
				case MGA_OP_MAX:  cukern_reduceScalar<MGA_OP_MAX><<<gridsize, blocksize>>>(in->devicePtr[i], blockValues[i], in->partNumel[i]); break;
				case MGA_OP_MIN:  cukern_reduceScalar<MGA_OP_MIN><<<gridsize, blocksize>>>(in->devicePtr[i], blockValues[i], in->partNumel[i]); break;
				}
		returnCode = CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, in, i, (const char *)"directionalMaxFinder(phi)");
		if(returnCode != SUCCESSFUL) break;
	}

	if(returnCode != SUCCESSFUL) return returnCode;

	// Since we get only 32*nGPUs elements back, not worth another kernel invocation
	double result = cpu_reduceInitValue(operate);
	int devCount = 0; // track which partition we're getting results from

	for(devCount = 0; devCount < in->nGPUs; devCount++) {
		hipSetDevice(in->deviceID[devCount]);
		returnCode = CHECK_CUDA_ERROR((const char *)"hipSetDevice()");
		if(returnCode != SUCCESSFUL) break;

		hipDeviceSynchronize(); // FIXME: can use less restrictive form here?
		returnCode = CHECK_CUDA_ERROR((const char *)"hipDeviceSynchronize()");
		if(returnCode != SUCCESSFUL) break;

		for(i = 0; i < gridsize.x; i++)
			result = cpu_reducePair(result, blockValues[devCount][i], operate);

		hipHostFree(blockValues[devCount]);
		returnCode = CHECK_CUDA_ERROR((const char *)"hipHostFree");
		if(returnCode != SUCCESSFUL) break;
	}

	scalar[0] = result;

	return returnCode;
}

/* The global reduction function accepts in addition a topology: After using the GPU to find
 * the local reduction, uses MPI to find the global value & stores it to scalar[0] for all
 * ranks participating in topology->comm.
 */
int MGA_globalReduceScalar(MGArray *in, double *scalar, MGAReductionOperator operate, ParallelTopology *topology)
{
	double nodeValue;
	int returnCode = MGA_localReduceScalar(in, &nodeValue, operate);
	if(CHECK_IMOGEN_ERROR(returnCode) != SUCCESSFUL) return returnCode;

	if(topology != NULL) {

		/* If parallel, now invoke MPI_Allreduce as well */
		MPI_Comm commune = MPI_Comm_f2c(topology->comm);
		int r0; MPI_Comm_rank(commune, &r0);
		int N; MPI_Comm_size(commune, &N);

		double globalValue;
		/* Perform the reduce */
		MPI_Allreduce((void *)&nodeValue, (void *)&globalValue, 1, MPI_DOUBLE, MGAReductionOperator_mga2mpi(operate), commune);

		scalar[0] = globalValue;
	} else {
		scalar[0] = nodeValue;
	}

	return SUCCESSFUL;
}

template <MGAReductionOperator OPERATION>
__global__ void cukern_ReduceX(double *phi, double *r, int nx);
template <MGAReductionOperator OPERATION>
__global__ void cukern_ReduceY(double *phi, double *r, int nx, int ny, int nz);
template <MGAReductionOperator OPERATION>
__global__ void cukern_ReduceZ(double *phi, double *r, int nx, int ny, int nz);

/* MGA_partitionReduceDimension is only meant to be called by MGA_localReduceDimension */
int MGA_partitionReduceDimension(MGArray *in, MGArray *out, MGAReductionOperator operate, int dir, int partition)
{
	int sub[6];
	calcPartitionExtent(in, partition, &sub[0]);

	dim3 blk, grid;

	hipSetDevice(in->deviceID[partition]);

	/* If the partition already has size 1, just copy input to output. */
	if(sub[2+dir] == 1) {
		hipMemcpyAsync(out->devicePtr[partition], in->devicePtr[partition], sizeof(double)*in->partNumel[partition], hipMemcpyDeviceToDevice);
		return CHECK_CUDA_ERROR((const char *)"partition reduce shortcircuit via memcpy");
	}

	switch(dir) {
	case 1: {
		blk = makeDim3(32,1,1);
		grid = makeDim3(sub[4], sub[5], 1);
		hipSetDevice(in->deviceID[partition]);
		switch(operate) {
		case MGA_OP_SUM:  cukern_ReduceX<MGA_OP_SUM> <<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3]); break;
		case MGA_OP_PROD: cukern_ReduceX<MGA_OP_PROD><<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3]); break;
		case MGA_OP_MAX:  cukern_ReduceX<MGA_OP_MAX> <<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3]); break;
		case MGA_OP_MIN:  cukern_ReduceX<MGA_OP_MIN> <<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3]); break;
		}
	} break;
	case 2: {
		blk = makeDim3(32, 8, 1);
		grid = makeDim3(ROUNDUPTO(sub[3],32)/32, ROUNDUPTO(sub[5],blk.y)/blk.y, 1);
		hipSetDevice(in->deviceID[partition]);
		switch(operate) {
		case MGA_OP_SUM:  cukern_ReduceY<MGA_OP_SUM> <<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3],sub[4],sub[5]); break;
		case MGA_OP_PROD: cukern_ReduceY<MGA_OP_PROD><<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3],sub[4],sub[5]); break;
		case MGA_OP_MAX:  cukern_ReduceY<MGA_OP_MAX> <<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3],sub[4],sub[5]); break;
		case MGA_OP_MIN:  cukern_ReduceY<MGA_OP_MIN> <<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3],sub[4],sub[5]); break;
		}
	} break;
	case 3: {
		blk = makeDim3(16, 16, 1);
		grid = makeDim3(ROUNDUPTO(sub[3],blk.x)/blk.x, ROUNDUPTO(sub[4],blk.y)/blk.y, 1);
		hipSetDevice(in->deviceID[partition]);
		switch(operate) {
		case MGA_OP_SUM:  cukern_ReduceZ<MGA_OP_SUM> <<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3],sub[4],sub[5]); break;
		case MGA_OP_PROD: cukern_ReduceZ<MGA_OP_PROD><<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3],sub[4],sub[5]); break;
		case MGA_OP_MAX:  cukern_ReduceZ<MGA_OP_MAX> <<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3],sub[4],sub[5]); break;
		case MGA_OP_MIN:  cukern_ReduceZ<MGA_OP_MIN> <<<blk, grid>>>(in->devicePtr[partition], out->devicePtr[partition], sub[3],sub[4],sub[5]); break;
		}
	} break;
	}

	return CHECK_CUDA_LAUNCH_ERROR(grid, blk, in, dir, (const char *)"Simple partition reduction function");

}

/* Invoke with blocks of 32 threads, and an [NY NZ 1] grid:
 * given size(phi) = [nx, gridDim.x, gridDim.y]
 * and   size(r)   = [gridDim.x, gridDim.y],
 * does
 *    r(blockIdx.x,blockIdx.y) <- OPERATION(phi(:,blockIdx.x,blockIdx.y)
 */
template <MGAReductionOperator OPERATION>
__global__ void cukern_ReduceX(double *phi, double *r, int nx)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int z = blockIdx.y;
	int ny = gridDim.x;

	__shared__ double W[32];

	double Q = cukern_reduceInitValue(OPERATION);

	if(x >= nx) return;

	phi += x + nx*(y+ny*z);

	while(x < nx) {
		Q = cukern_reducePair(Q, *phi, OPERATION);
		x += 32;
		phi += 32;
	}

	W[threadIdx.x] = Q;

	x = 16;
	int tix = threadIdx.x;
	if(tix >= x) return;

	/* This is relevant if in future block > 1 warp */
	while(x > 16) {
		if(tix >= x) return;
		__syncthreads();
		if(W[tix+x] > W[tix]) W[tix] = W[tix+x];
	        x=x/2;
	}

	__syncthreads();

	// We have one halfwarp (16 threads) remaining, proceed synchronously on assumption of warp-level SIMD synchronicity
	if(W[tix+16] > W[tix]) W[tix] = W[tix+16]; if(tix >= 8) return;
	if(W[tix+8] > W[tix]) W[tix] = W[tix+8]; if(tix >= 4) return;
	if(W[tix+4] > W[tix]) W[tix] = W[tix+4]; if(tix >= 2) return;
	if(W[tix+2] > W[tix]) W[tix] = W[tix+2]; if(tix) return;

	/* last guy out, please turn off the lights */
	r[y+ny*z] = (W[1] > W[0]) ? W[1] : W[0];
}

/* Invoke with blocks of [32 A] threads and [ceil(NX/32), ceil(NZ/A) 1 1 ] grid
 * given size(phi) = [nx ny nz]
 * and   size(r)   = [nx nz]
 * does
 *     r(x, z) <- OPERATION(phi(x,:,z))
 */
template <MGAReductionOperator OPERATION>
__global__ void cukern_ReduceY(double *phi, double *r, int nx, int ny, int nz)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = 0;
	int z = threadIdx.y + blockIdx.y*blockDim.y;

	if(x >= nx) return;
	if(z >= nz) return;
	phi += x + nx*ny*z;

	double Q = cukern_reduceInitValue(OPERATION);
	while(y < ny) {
		Q = cukern_reducePair(Q, phi[nx*ny], OPERATION);
		y++;
	}

	r[x+nx*z] = Q;
}

/* Invoke with [A B 1] block and [C D 1] grid such that
 *   AC >= nx
 *   BD >= ny
 * Given size(phi) = [nx ny nz] and size(r) = [nx ny 1], does
 *    r(x,y) <- OPERATION(phi(x,y,:))
 */
template <MGAReductionOperator OPERATION>
__global__ void cukern_ReduceZ(double *phi, double *r, int nx, int ny, int nz)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	if(x >= nx) return;
	if(y >= ny) return;
	phi += x + nx*y;
	r   += x + nx*y;

	double Q = cukern_reduceInitValue(OPERATION);

	int z = 0;
	int step = nx*ny;
	for(z = 0; z < nz; z++) {
		Q = cukern_reducePair(Q, *phi, OPERATION);
		phi += step;
	}

	*r = Q;
}

/* MGA_localReduceDimension operates such that
 *    out[...,dir=0,...] = REDUCE(in[...,:,...])
 *    out.dim[dir-1] will equal 1.
 * i.e. reduction to scalar is done to every vector of elements in the 'dir' direction.
 *
 * If the in->partitionDir == dir, the answer stored to partition 'partitionOnto.'
 *    If this is the case and 'redistribute' is set, this data is copied to all other partitions before return.
 * If in->partitionDir != dir, 'partitionOnto' and 'redistribute' are irrelevant.
 *
 * If out[0] is nonnull and the correct size, its contents are overwritten with the output array.
 * If out[0] is nonnull but is an incorrect size, ERROR_INVALID_ARGS occurs.
 * If out[0] is null, an output array of the correct size is allocated. */
int MGA_localReduceDimension(MGArray *in, MGArray **out, MGAReductionOperator operate, int dir, int partitionOnto, int redistribute)
{
	int returnCode = SUCCESSFUL;
	int i;

	MGArray clone = *in;

	// Determine what size the reduced array should be
	if(dir == in->partitionDir) {
		clone.haloSize = 0;
		clone.dim[dir-1] = in->nGPUs; // to flatten
	} else {
		clone.haloSize = 0;
		clone.dim[dir-1] = 1; //
	}

	// check or allocate
	if(out[0] != NULL) {
		if(out[0]->dim[0] != clone.dim[0]) returnCode = ERROR_INVALID_ARGS;
		if(out[0]->dim[1] != clone.dim[1]) returnCode = ERROR_INVALID_ARGS;
		if(out[0]->dim[2] != clone.dim[2]) returnCode = ERROR_INVALID_ARGS;
		if(returnCode != SUCCESSFUL) {
			PRINT_FAULT_HEADER;
			printf((const char *)"out[0] was not null, but the passed MGArray** is of inappropriate dimensions.\nCannot safely free it & overwrite: Must return error.\n");
			PRINT_FAULT_FOOTER;
			return returnCode;
		}
	} else {
		 returnCode = MGA_allocArrays(&out[0], 1, &clone);
		 if(CHECK_IMOGEN_ERROR(returnCode) != SUCCESSFUL) { return returnCode; }
	}

	// Call per-partition reductions
	for(i = 0; i < in->nGPUs; i++) {
		returnCode = MGA_partitionReduceDimension(in, out[0], operate, dir, i);
		if(returnCode != SUCCESSFUL) { return CHECK_IMOGEN_ERROR(returnCode); }
	}

	if(dir == in->partitionDir) {
		// reduce across partitions
		returnCode = MGA_reduceAcrossDevices(out[0], operate, redistribute);
	}

	return CHECK_IMOGEN_ERROR(returnCode);
}

/* First uses MGA_localReduceDimension to perform an accelerated reduce locally.
 * Then the result is copied to host memory and the MPI communicator in topology->dimcomm
 * corresponding to the direction is used to find the reduction across nodes,
 * then results transferred back to the GPU.
 */
int MGA_globalReduceDimension(MGArray *in, MGArray **out, MGAReductionOperator operate, int dir, int partitionOnto, int redistribute, ParallelTopology *topology)
{
	int returnCode = SUCCESSFUL;

	MGArray clone = *in;

	// Determine what size the reduced array should be
	if(dir == in->partitionDir) {
		clone.haloSize = 0;
		clone.dim[dir-1] = in->nGPUs; // flatten
	} else {
		clone.haloSize = 0;
		clone.dim[dir-1] = 1;
	}

	// check or allocate
	if(out[0] != NULL) {
		if(out[0]->dim[0] != clone.dim[0]) returnCode = ERROR_INVALID_ARGS;
		if(out[0]->dim[1] != clone.dim[1]) returnCode = ERROR_INVALID_ARGS;
		if(out[0]->dim[2] != clone.dim[2]) returnCode = ERROR_INVALID_ARGS;
		if(returnCode != SUCCESSFUL) {
			PRINT_FAULT_HEADER;
			printf((const char *)"out[0] was not null, but the passed MGArray** is of inappropriate dimensions.\nCannot safely free it & overwrite: Must return error.\n");
			PRINT_FAULT_FOOTER;
			return returnCode;
		}
	} else {
		returnCode = MGA_allocArrays(&out[0], 1, &clone);
		if(CHECK_IMOGEN_ERROR(returnCode) != SUCCESSFUL) { return returnCode; }
	}

	/* All ranks flatten to 1D in reduce dimension in parallel */
	returnCode = MGA_localReduceDimension(in, out, operate, dir, partitionOnto, 0);
	if(returnCode != SUCCESSFUL) { return CHECK_IMOGEN_ERROR(returnCode); }

	/* Skip parallel reduction if no topology is forthcoming */
	if(topology != NULL) {

		/* Reverse silly memory ordering */
		//int d = dir - 1;
		//int dmax = topology->nproc[d];

		MPI_Comm commune = MPI_Comm_f2c(topology->comm);
		int r0; MPI_Comm_rank(commune, &r0);

		double *readBuf = NULL;
		if(dir == out[0]->partitionDir) {
		    returnCode = MGA_downloadArrayToCPU(out[0], &readBuf, partitionOnto);
		} else {
			returnCode = MGA_downloadArrayToCPU(out[0], &readBuf, -1);
		}
		if(returnCode != SUCCESSFUL) { return CHECK_IMOGEN_ERROR(returnCode); }

		double *writeBuf= (double *)malloc(out[0]->numel*sizeof(double));
		if(writeBuf == NULL) {
			PRINT_FAULT_HEADER;
			printf((const char *)"Failed to allocate write buffer memory!\n");
			PRINT_FAULT_FOOTER;
			return ERROR_NULL_POINTER;
		}

		int numToReduce = (dir == out[0]->partitionDir) ? out[0]->partNumel[partitionOnto] : out[0]->numel;
		MPI_Comm dircom = MPI_Comm_f2c(topology->dimcomm[dir-1]);

		MPI_Allreduce((void *)readBuf, (void *)writeBuf, numToReduce, MPI_DOUBLE, MGAReductionOperator_mga2mpi(operate), dircom);

		MPI_Barrier(dircom);

		int upPart = (dir == out[0]->partitionDir) ? partitionOnto : -1;
		returnCode = MGA_uploadArrayToGPU(writeBuf, out[0], upPart);

		if(returnCode != SUCCESSFUL) return CHECK_IMOGEN_ERROR(returnCode);

		if(redistribute && (dir == out[0]->partitionDir)) returnCode = MGA_distributeArrayClones(out[0], partitionOnto);

		free(readBuf); free(writeBuf);
		// This leaks if we encounter an error
		// But if that's the case we're crashing anyway...

		return CHECK_IMOGEN_ERROR(returnCode);
	} else {
		return SUCCESSFUL;
	}

}


template<MGAReductionOperator OP>
__global__ void cukern_TwoElementwiseReduce(double *a, double *b, int numel);
template<MGAReductionOperator OP>
__global__ void cudaClonedReducerQuad(double *a, double *b, double *c, double *d, int numel);

/* Requiring that each partition have equal # of elements, computes
 * a->devicePtr[0][i] = REDUCTION(a->devicePtr[0][i], a->devicePtr[1][i], ..., a->devicePtr[a->nGPUs][i])
 * for i in 0 to a->partNumel[0].
 *
 * if redistribute == 1, a->devicePtr[0] data is copied to partitions 1 through a->nGPUs as well. */
int MGA_reduceAcrossDevices(MGArray *a, MGAReductionOperator operate, int redistribute)
{
	int i;

	// Check that this operation is acceptable
	for(i = 1; i < a->nGPUs; i++) {
		if(a->partNumel[i] != a->partNumel[0]) return ERROR_INVALID_ARGS;
	}

	int eachPartSize = a->partNumel[0];

	int returnCode = SUCCESSFUL;

	dim3 gridsize; gridsize.x = 32; gridsize.y = gridsize.z = 1;
	dim3 blocksize; blocksize.x = 256; blocksize.y = blocksize.z = 1;

	double *B; double *C;

	switch(a->nGPUs) {
	case 1: break; // Well this was a waste of time
	case 2: // reduce(A,B)->A
		hipSetDevice(a->deviceID[0]);
		returnCode = CHECK_CUDA_ERROR((const char *)"hipSetDevice()");
		if(returnCode != SUCCESSFUL) break;
		hipMalloc((void **)&B, eachPartSize*sizeof(double));
		returnCode = CHECK_CUDA_ERROR((const char *)"hipMalloc()");
		if(returnCode != SUCCESSFUL) break;
		hipMemcpy((void *)B, (void*)a->devicePtr[1], eachPartSize*sizeof(double), hipMemcpyDeviceToDevice);
		returnCode = CHECK_CUDA_ERROR((const char *)"hipMalloc()");
		if(returnCode != SUCCESSFUL) break;

		switch(operate) {
		case MGA_OP_SUM: cukern_TwoElementwiseReduce<MGA_OP_SUM><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_PROD: cukern_TwoElementwiseReduce<MGA_OP_PROD><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_MIN: cukern_TwoElementwiseReduce<MGA_OP_MIN><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_MAX: cukern_TwoElementwiseReduce<MGA_OP_MAX><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		}

		returnCode = CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, a, 2, "clone reduction for 2 GPUs");
		if(returnCode != SUCCESSFUL) break;
		hipFree(B);
		returnCode = CHECK_CUDA_ERROR((const char *)"hipFree()");
		if(returnCode != SUCCESSFUL) break;
		break;
	case 3: // reduce(A,B)->A; reduce(A, C)->A
		hipSetDevice(a->deviceID[0]);
		returnCode = CHECK_CUDA_ERROR((const char *)"hipSetDevice()");
		if(returnCode != SUCCESSFUL) break;
		hipMalloc((void **)&B, eachPartSize*sizeof(double));
		returnCode = CHECK_CUDA_ERROR((const char *)"cuda malloc");
		if(returnCode != SUCCESSFUL) break;
		hipMalloc((void **)&C, eachPartSize*sizeof(double));
		returnCode = CHECK_CUDA_ERROR((const char *)"cuda malloc");
		if(returnCode != SUCCESSFUL) break;

		hipMemcpy((void *)B, (void *)a->devicePtr[1], eachPartSize*sizeof(double), hipMemcpyDeviceToDevice);
		returnCode = CHECK_CUDA_ERROR((const char *)"cuda memcpy");
		if(returnCode != SUCCESSFUL) break;
		hipMemcpy((void *)C, (void *)a->devicePtr[2], eachPartSize*sizeof(double), hipMemcpyDeviceToDevice);
		returnCode = CHECK_CUDA_ERROR((const char *)"cuda memcpy");
		if(returnCode != SUCCESSFUL) break;

		switch(operate) {
		case MGA_OP_SUM: cukern_TwoElementwiseReduce<MGA_OP_SUM><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_PROD: cukern_TwoElementwiseReduce<MGA_OP_PROD><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_MIN: cukern_TwoElementwiseReduce<MGA_OP_MIN><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_MAX: cukern_TwoElementwiseReduce<MGA_OP_MAX><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		}

		returnCode = CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, a, 2, (const char *)"clone reduction for 3 GPUs, first call");
		if(returnCode != SUCCESSFUL) break;

		switch(operate) {
		case MGA_OP_SUM: cukern_TwoElementwiseReduce<MGA_OP_SUM><<<32, 256>>>(a->devicePtr[0], C, eachPartSize); break;
		case MGA_OP_PROD: cukern_TwoElementwiseReduce<MGA_OP_PROD><<<32, 256>>>(a->devicePtr[0], C, eachPartSize); break;
		case MGA_OP_MIN: cukern_TwoElementwiseReduce<MGA_OP_MIN><<<32, 256>>>(a->devicePtr[0], C, eachPartSize); break;
		case MGA_OP_MAX: cukern_TwoElementwiseReduce<MGA_OP_MAX><<<32, 256>>>(a->devicePtr[0], C, eachPartSize); break;
		}

		returnCode = CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, a, 2, (const char *)"clone reduction for 3 GPUs, second call");
		if(returnCode != SUCCESSFUL) break;

		hipFree(B);
		returnCode = CHECK_CUDA_ERROR((const char *)"hipFree");
		if(returnCode != SUCCESSFUL) break;
		hipFree(C);
		returnCode = CHECK_CUDA_ERROR((const char *)"hipFree");
		if(returnCode != SUCCESSFUL) break;

		break;
	case 4: // {reduce(A,B)->A, reduce(C,D)->C}; reduce(A,C)->A
		// FIXME: This is broken right now...
//		mexErrMsgTxt((const char *)"This is broken soz.");

		// On device 0, allocate storage for device 1 and copy device 1 partition to device 0
		hipSetDevice(a->deviceID[0]);
		CHECK_CUDA_ERROR((const char *)"hipSetDevice()");
		hipMalloc((void **)&B, eachPartSize*sizeof(double));
		CHECK_CUDA_ERROR((const char *)"hipMalloc");
		hipMemcpyAsync((void *)B, (void *)a->devicePtr[1], eachPartSize*sizeof(double), hipMemcpyDeviceToDevice);
		returnCode = CHECK_CUDA_ERROR((const char *)"cuda memcpy");
		if(returnCode != SUCCESSFUL) break;

		// Launch (A,B)->A reduction on device 0
		switch(operate) {
		case MGA_OP_SUM: cukern_TwoElementwiseReduce<MGA_OP_SUM><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_PROD: cukern_TwoElementwiseReduce<MGA_OP_PROD><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_MIN: cukern_TwoElementwiseReduce<MGA_OP_MIN><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_MAX: cukern_TwoElementwiseReduce<MGA_OP_MAX><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		}
		returnCode = CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, a, 2, "clone reduction for 4 GPUs, first call (A,B)->A");
		if(returnCode != SUCCESSFUL) break;

		// On device 2, allocate storage for device 3 and copy device 3 partition to device 2
		hipSetDevice(a->deviceID[2]);
		CHECK_CUDA_ERROR((const char *)"hipSetDevice()");
		hipMalloc((void **)&C, eachPartSize*sizeof(double));
		CHECK_CUDA_ERROR((const char *)"hipMalloc");
		hipMemcpyAsync((void *)C, (void *)a->devicePtr[3], eachPartSize*sizeof(double), hipMemcpyDeviceToDevice);
		returnCode = CHECK_CUDA_ERROR("cuda memcpy");
		if(returnCode != SUCCESSFUL) break;

		// Launch (C,D)->C reduction on device 2
		switch(operate) {
		case MGA_OP_SUM: cukern_TwoElementwiseReduce<MGA_OP_SUM><<<32, 256>>>(a->devicePtr[2], C, eachPartSize); break;
		case MGA_OP_PROD: cukern_TwoElementwiseReduce<MGA_OP_PROD><<<32, 256>>>(a->devicePtr[2], C, eachPartSize); break;
		case MGA_OP_MIN: cukern_TwoElementwiseReduce<MGA_OP_MIN><<<32, 256>>>(a->devicePtr[2], C, eachPartSize); break;
		case MGA_OP_MAX: cukern_TwoElementwiseReduce<MGA_OP_MAX><<<32, 256>>>(a->devicePtr[2], C, eachPartSize); break;
		}
		returnCode = CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, a, 2, "clone reduction for 4 GPUs, second call (C,D)->C");
		if(returnCode != SUCCESSFUL) break;

		// We are going to make this work the simple way
		// We need to be sure device 2 is done before we copy from device 2 to device 0
		hipDeviceSynchronize();

		// Copy C -> A for the final reduction
		hipSetDevice(a->deviceID[0]);

		hipMemcpyAsync((void *)B, (void *)a->devicePtr[2], eachPartSize*sizeof(double), hipMemcpyDeviceToDevice);
		returnCode = CHECK_CUDA_ERROR("cuda memcpy");
		if(returnCode != SUCCESSFUL) break;

		switch(operate) {
		case MGA_OP_SUM: cukern_TwoElementwiseReduce<MGA_OP_SUM><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_PROD: cukern_TwoElementwiseReduce<MGA_OP_PROD><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_MIN: cukern_TwoElementwiseReduce<MGA_OP_MIN><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		case MGA_OP_MAX: cukern_TwoElementwiseReduce<MGA_OP_MAX><<<32, 256>>>(a->devicePtr[0], B, eachPartSize); break;
		}
		returnCode = CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, a, 2, "clone reduction for 4 GPUs, 3rd call (A,C)->A");
		if(returnCode != SUCCESSFUL) break;

		hipFree(B);
		returnCode = CHECK_CUDA_ERROR("hipFree");
		if(returnCode != SUCCESSFUL) break;

		hipSetDevice(a->deviceID[2]);
		hipFree(C);
		returnCode = CHECK_CUDA_ERROR("hipFree");
		if(returnCode != SUCCESSFUL) break;

		break;
	default: return -1;
	}

	if(returnCode != SUCCESSFUL) return CHECK_IMOGEN_ERROR(returnCode);

	if(redistribute)
		returnCode = MGA_distributeArrayClones(a, 0);

	return CHECK_IMOGEN_ERROR(returnCode);
}

/* If partition sizes are equal (as typical e.g. post-reduction),
 * copies the array on partition partitionFrom to the others resulting in
 * identical copies on each device.
 */
int MGA_distributeArrayClones(MGArray *cloned, int partitionFrom)
{
	int returnCode = SUCCESSFUL;
	int j;

	for(j = 1; j < cloned->nGPUs; j++) {
		if(cloned->partNumel[j] != cloned->partNumel[0]) return ERROR_INVALID_ARGS;
	}

	hipSetDevice(cloned->deviceID[partitionFrom]);

	for(j = 0; j < cloned->nGPUs; j++) {
		if(j == partitionFrom) continue;

		hipMemcpyAsync(cloned->devicePtr[j], cloned->devicePtr[partitionFrom], sizeof(double)*cloned->partNumel[partitionFrom], hipMemcpyDeviceToDevice);
		returnCode = CHECK_CUDA_ERROR("MGA_distributeArrayClones");
		if(returnCode != SUCCESSFUL) break;
	}

	return CHECK_IMOGEN_ERROR(returnCode);
}

/* Returns true IFF arrays a and b have identical memory layouts,
 * (= same exterior halo setting, haloSize, dim[i], & device IDs)
 * This means that all same-cell indexing ops will be identical &
 * may imply combining, or not having to reallocate, temp storage
 * Does NOT check numSlabs */
int MGA_arraysAreIdenticallyShaped(MGArray *a, MGArray *b)
{
	if((a == NULL) || (b == NULL)) return 0;

	if(a->addExteriorHalo != b->addExteriorHalo) return 0; // same halo settings...
	if(a->nGPUs != b->nGPUs) return 0; // same gpu count...
	if(a->haloSize != b->haloSize) return 0; // same halos size...
	int i;
	for(i = 0; i < 3; i++) { if(a->dim[i] != b->dim[i]) return 0; } // same external dimension...
	for(i = 0; i < a->nGPUs; i++) {
		if(a->deviceID[i] != b->deviceID[i]) return 0; // same device
	}

	return 1;
}

template<MGAReductionOperator OP>
__global__ void cukern_TwoElementwiseReduce(double *a, double *b, int numel)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int step_i = blockDim.x*gridDim.x;

	for(; i < numel; i+= step_i) {
		a[i] = cukern_reducePair(a[i], b[i], OP);
	}
}

#define LEFT_SIDE 0
#define RIGHT_SIDE 1
#define HALO_READ 0
#define HALO_WRITE 1

int MGA_exchangeLocalHalos_identical(MGArray *a, int nArrays);

/* Synchronizes the halo regions between data partitions of a[0] to a[n-1].
 * Does nothing if a[i].haloSize == 0 or a[i].nGPUs == 1.
 * returns error if something failed, or SUCCESSFUL */
int MGA_exchangeLocalHalos(MGArray *a, int n)
{
	int i, j, jn, jp;
	dim3 blocksize, gridsize;
	int returnCode = SUCCESSFUL;

	// Checking this @ runtime is as good as free but may save as many as n*nGPUs
	// cudaMallocs and cudaFrees, potentially saving entire msecs.
	int arrayGeometryIdentical = 1;
        for(j = 1; j < n; j++) {
                if(MGA_arraysAreIdenticallyShaped(a, a+j) == 0) {
                        arrayGeometryIdentical = 0;
                }
        }

    //if(arrayGeometryIdentical) {
     // 	return CHECK_IMOGEN_ERROR(MGA_exchangeLocalHalos_identical(a, n));
    //}

    double *buffs[a->nGPUs * 4];

	for(i = 0; i < n; i++) {
		// Can't do this if there are no halos
		if(a->haloSize == 0) { break; }
		// Or there's only one partition to begin with
		if(a->nGPUs == 1) { break; }

		int sub[6];

		calcPartitionExtent(a, 0, &sub[0]);

		// Acquire sufficient RW linear buffers to R and W both sides
		int numTransverse = a->partNumel[0] / sub[2+a->partitionDir];
		int numHalo = a->haloSize * numTransverse;

		if(a->partitionDir != PARTITION_Z) {
			if((i == 0) || (arrayGeometryIdentical == 0)) {
			for(j = 0; j < a->nGPUs; j++) {
				hipSetDevice(a->deviceID[j]);
				CHECK_CUDA_ERROR("hipSetDevice()");
				hipMalloc((void **)&buffs[4*j], 4*numHalo*sizeof(double));
				returnCode = CHECK_CUDA_ERROR("hipMalloc");
				if(returnCode != SUCCESSFUL) break;
				buffs[4*j+1] = buffs[4*j] + 1*numHalo;
				buffs[4*j+2] = buffs[4*j] + 2*numHalo;
				buffs[4*j+3] = buffs[4*j] + 3*numHalo;
			}
			}

			// Fetch current partition's halo to linear strips, letting jn denote next and jp denote previous
			for(j = 0; j < a->nGPUs; j++) {
				jn = (j+1) % a->nGPUs;
				jp = (j - 1 + a->nGPUs) % a->nGPUs;

				// If addExteriorHalo is set, we behave circularly
				// This is appropriate if e.g. we have only one MPI rank in the partitioned direction with a circular BC

				// If there are N>1 MPI ranks in the U direction and we are partitioned in U,
				// We do not handle these boundaries & leave them to MPI (cudaHaloExchange)

				// At first glance full-circular here isn't a problem (after all, the MPI exchange will just overwrite
				// this, right?). However, IF cudaHaloExchange is involved, then outside-MGA things (i.e.
				// Matlab) will be aware of the halo because it was added by GeometryManager, not MGA.
				// Then our use of the halo here will corrupt visible data.

				// In particular, it will corrupt the calculation of boundary conditions!
				if(a->addExteriorHalo || (j > 0)) {
					returnCode = MGA_partitionHaloToLinear(a, j, a->partitionDir, LEFT_SIDE, HALO_READ, a->haloSize, &buffs[4*j+0]);
					if(returnCode != SUCCESSFUL) break;
				}
				if(a->addExteriorHalo || (j < (a->nGPUs-1))) {
					returnCode = MGA_partitionHaloToLinear(a, j, a->partitionDir, RIGHT_SIDE, HALO_READ, a->haloSize, &buffs[4*j+1]);
					if(returnCode != SUCCESSFUL) break;
				}
			}

//MGA_sledgehammerSequentialize(a);
			// Transfer linear strips
			for(j = 0; j < a->nGPUs; j++) {
				hipSetDevice(a->deviceID[j]);
				CHECK_CUDA_ERROR("hipSetDevice()");

				jn = (j+1) % a->nGPUs; jp = (j - 1 + a->nGPUs) % a->nGPUs;
				if(a->addExteriorHalo || (j > 0)) {
					hipMemcpyPeerAsync(buffs[4*jp+3], a->deviceID[jp], buffs[4*j], a->deviceID[j], numHalo * sizeof(double));
					returnCode = CHECK_CUDA_ERROR("hipMemcpyPeer");
					if(returnCode != SUCCESSFUL) break;
				}
				if(a->addExteriorHalo || (j < (a->nGPUs-1))) {
					hipMemcpyPeerAsync(buffs[4*jn+2], a->deviceID[jn], buffs[4*j+1], a->deviceID[j], numHalo * sizeof(double));
					returnCode = CHECK_CUDA_ERROR("hipMemcpyPeer");
					if(returnCode != SUCCESSFUL) break;
				}

			}
//MGA_sledgehammerSequentialize(a);
			// Dump the strips back to halo
			for(j = 0; j < a->nGPUs; j++) {
				jn = (j+1) % a->nGPUs; jp = (j - 1 + a->nGPUs) % a->nGPUs;
				if(a->addExteriorHalo || (j > 0)) {
					returnCode = MGA_partitionHaloToLinear(a, jp, a->partitionDir, RIGHT_SIDE, HALO_WRITE, a->haloSize, &buffs[4*jp+3]);
					if(returnCode != SUCCESSFUL) break;
				}
				if(a->addExteriorHalo || (j < (a->nGPUs-1))) {
					returnCode = MGA_partitionHaloToLinear(a, jn, a->partitionDir, LEFT_SIDE, HALO_WRITE, a->haloSize, &buffs[4*jn+2]);
					if(returnCode != SUCCESSFUL) break;
				}
			}

			// Let go of temp memory
			if((i == n-1) || (arrayGeometryIdentical == 0)) {
			for(j = 0; j < a->nGPUs; j++) {
				hipSetDevice(a->deviceID[j]);
				CHECK_CUDA_ERROR("hipSetDevice");
				hipFree(buffs[4*j]);
				returnCode = CHECK_CUDA_ERROR("hipFree");
				if(returnCode != SUCCESSFUL) break;
			}
			}
			if(returnCode != SUCCESSFUL) break;

		} else {
			/* Z halos are delightful, we simply copy some already-linearly-contiguous blocks
			 * of memory back and forth. the partition halo call would *work* but we can short-circuit
			 * pointless copying this way.
			 */

			for(j = 0; j < a->nGPUs; j++) {
				hipSetDevice(a->deviceID[j]);
				calcPartitionExtent(a, j, sub);
				int subp[6];

				jn = (j+1) % a->nGPUs; // Next partition
				jp = (j-1+a->nGPUs) % a->nGPUs; // previous partition

				calcPartitionExtent(a, jp, subp);

				size_t halotile = a->dim[0]*a->dim[1];
				size_t byteblock = halotile*a->haloSize*sizeof(double);

				//size_t L_halo = (subp[5] - a->haloSize)*halotile;
				size_t L_src  = (subp[5]-2*a->haloSize)*halotile;
				size_t R_halo = (sub[5] - a->haloSize)*halotile;
				//size_t R_src  = (sub[5] -2*a->haloSize)*halotile;

				// If we have an exterior halo, or this is NOT the last GPU,
				// fill the +side halo of THIS partition (j) with data from the -side of the next one (jn):
				if(a->addExteriorHalo || (j < (a->nGPUs-1))) {
					// Fill right halo with left's source
					hipMemcpyAsync((void *)(a->devicePtr[j]+R_halo),
						   (void *)(a->devicePtr[jn]+halotile*a->haloSize), byteblock, hipMemcpyDeviceToDevice);
					returnCode = CHECK_CUDA_ERROR("hipMemcpy");
					if(returnCode != SUCCESSFUL) break;
				}

				// If we have an exterior halo (then always), or this is NOT the FIRST gpu,
				// fill the -side halo of this partition (j) with data from the +side of the previous one (jp):
				if(a->addExteriorHalo || (j > 0)) {
					// Fill left halo with right's source
					hipMemcpyAsync((void *)(a->devicePtr[j]),
							(void *)(a->devicePtr[jp]+L_src), byteblock, hipMemcpyDeviceToDevice);
					returnCode = CHECK_CUDA_ERROR("hipMemcpy");
					if(returnCode != SUCCESSFUL) break;\
				} 

				hipDeviceSynchronize();

			}

		}

		a++;

	}

	return CHECK_IMOGEN_ERROR(returnCode);
}

/* This is called to do the local exchange if MGA_arraysAreIdenticallyShaped(a, a+i)
 * is true for 0 <= i < n (i.e. all arrays are the same). Allocations are done only
 * once, and all memcopies are aggreggated into a single move which yields much
 * improved IO throughput.
 */
int MGA_exchangeLocalHalos_identical(MGArray *a, int nArrays)
{
	int i, j, jn, jp;
	dim3 blocksize, gridsize;
	int returnCode = SUCCESSFUL;

	// Can't do this if there are no halos
	if(a->haloSize == 0) { return SUCCESSFUL; }
	// Or there's only one partition to begin with
	if(a->nGPUs == 1) { return SUCCESSFUL; }

	int sub[6];
	calcPartitionExtent(a, 0, &sub[0]);

	// Acquire sufficient RW linear buffers to R and W both sides
	int numTransverse = a->partNumel[0] / sub[2+a->partitionDir];
	int numHalo = a->haloSize * numTransverse;

	// We don't distinguish between Z- and not-Z partitioning here because we need
	// to aggregate all the data to be copied over pcie into one block anyway

	double *allocedLinear[a->nGPUs];
	double *buffA, *buffB;
	//total storage required: (numHalo * 4) * n

	int totHalo = numHalo * nArrays;

	for(j = 0; j < a->nGPUs; j++) {
		hipSetDevice(a->deviceID[j]);
		CHECK_CUDA_ERROR("hipSetDevice()");
		hipMalloc((void **)&allocedLinear[j], nArrays*4*numHalo*sizeof(double));
		returnCode = CHECK_CUDA_ERROR("hipMalloc");
		if(returnCode != SUCCESSFUL) break;
	}

	// All arrays fetch linear blocks
	for(i = 0; i < nArrays; i++) {
		// Fetch current partition's halo to linear strips, letting jn denote next and jp denote previous
		for(j = 0; j < a->nGPUs; j++) {
			buffA = allocedLinear[j] + (0*nArrays+i)*numHalo;
			buffB = allocedLinear[j] + (1*nArrays+i)*numHalo;

			if(a->addExteriorHalo || (j > 0)) {
				returnCode = MGA_partitionHaloToLinear(a, j, a->partitionDir, LEFT_SIDE, HALO_READ, a->haloSize, &buffA);
				if(returnCode != SUCCESSFUL) break;
			}
			if(a->addExteriorHalo || (j < (a->nGPUs-1))) {
				returnCode = MGA_partitionHaloToLinear(a, j, a->partitionDir, RIGHT_SIDE, HALO_READ, a->haloSize, &buffB);
				if(returnCode != SUCCESSFUL) break;
			}
		}
	}
	MGA_sledgehammerSequentialize(a);
	// Transfer linear strips
	for(j = 0; j < a->nGPUs; j++) {
		jn = (j+1) % a->nGPUs;
		jp = (j - 1 + a->nGPUs) % a->nGPUs;

		hipSetDevice(a->deviceID[j]);
		CHECK_CUDA_ERROR("hipSetDevice()");

		// Copy j's left side read to j-previous' right-side write
		if(a->addExteriorHalo || (j > 0)) {
			hipMemcpyPeerAsync(allocedLinear[jp]+3*totHalo, a->deviceID[jp], allocedLinear[j]+0*totHalo, a->deviceID[j], totHalo * sizeof(double));
			returnCode = CHECK_CUDA_ERROR("hipMemcpyPeer");
			if(returnCode != SUCCESSFUL) break;
		}
		// Copy j's right side read to j-previous' left-side write
		if(a->addExteriorHalo || (j < (a->nGPUs-1))) {
			hipMemcpyPeerAsync(allocedLinear[jn]+2*totHalo, a->deviceID[jn], allocedLinear[j]+1*totHalo, a->deviceID[j], totHalo * sizeof(double));
			returnCode = CHECK_CUDA_ERROR("hipMemcpyPeer");
			if(returnCode != SUCCESSFUL) break;
		}

	}
	MGA_sledgehammerSequentialize(a);
	for(i = 0; i < nArrays; i++) {
		// Dump the strips back to halo
		for(j = 0; j < a->nGPUs; j++) {
			jn = (j+1) % a->nGPUs;
			jp = (j - 1 + a->nGPUs) % a->nGPUs;

			buffA = allocedLinear[jp] + (3*nArrays+i)*numHalo;
			buffB = allocedLinear[jn] + (2*nArrays+i)*numHalo;

			if(a->addExteriorHalo || (j > 0)) {
				returnCode = MGA_partitionHaloToLinear(a, jp, a->partitionDir, RIGHT_SIDE, HALO_WRITE, a->haloSize, &buffA);
				if(returnCode != SUCCESSFUL) break;
			}
			if(a->addExteriorHalo || (j < (a->nGPUs-1))) {
				returnCode = MGA_partitionHaloToLinear(a, jn, a->partitionDir, LEFT_SIDE, HALO_WRITE, a->haloSize, &buffB);
				if(returnCode != SUCCESSFUL) break;
			}
		}

	}

	// Let go of temp memory
	for(j = 0; j < a->nGPUs; j++) {
		hipSetDevice(a->deviceID[j]);
		CHECK_CUDA_ERROR("hipSetDevice");
		hipFree(allocedLinear[j]);
		returnCode = CHECK_CUDA_ERROR("hipFree");
		if(returnCode != SUCCESSFUL) break;
	}

	return CHECK_IMOGEN_ERROR(returnCode);
}




int MGA_wholeFaceHaloNumel(MGArray *a, int direction, int h)
{
if(a == NULL) DROP_MEX_ERROR("In MGA_faceHaloNumel sanity checks: a is NULL!\n");

int q = 0;

if(a->partitionDir == direction) {
	q = MGA_partitionHaloNumel(a, 0, direction, h);
} else {
	int i;
	for(i = 0; i < a->nGPUs; i++) {
		q += MGA_partitionHaloNumel(a, i, direction, h);
	}
}

return q;

}

/* From the MGArray pointed to by 'a', will read to CPU (writehalo == 0) or upload to gpu
 * (writehalo == 1) the lowest-index (rightside == 0) or highest-index (rightside == 1)
 * face of 'a' in the 'direction' direction (x=1,y=2,z=3) from linear[0].
 * If linear[0] is null, allocates storage into it.
 * If linear[0] is not null, it must point to sufficient storage.
 *
 * FIXME: This routine is potentially dangerous when called to fetch halos for outside-MGA purposes
 * FIXME: Reason: it assumes that ranks A and B have identical partitioning, in which case the metadata
 * FIXME: associated with MGA_partitionHaloToLinear output will be the same (i.e. partition i on rank B
 * FIXME: will have the same size, halo size and index permutation as partition i on rank A)
 */
int MGA_wholeFaceToLinear(MGArray *a, int direction, int rightside, int writehalo, int h, double **linear)
{

	int returnCode = SUCCESSFUL;
	if(direction == a->partitionDir) {
		int part = 0;
		if(rightside) part = a->nGPUs - 1;

		returnCode = MGA_partitionHaloToLinear(a, part, direction, rightside, writehalo, h, linear);
	} else { // Fetch all halo partitions
		int q = 0;
		int ctr;
		for(ctr = 0; ctr < a->nGPUs; ctr++) {
			double *ptmp = linear[0] + q;
			returnCode = MGA_partitionHaloToLinear(a, ctr, direction, rightside, writehalo, h, &ptmp);
			if(returnCode != SUCCESSFUL) break;
			q += MGA_partitionHaloNumel(a, ctr, direction, a->haloSize);
		}
	}

	return CHECK_IMOGEN_ERROR(returnCode);
}


/* Determines how many linear memory elements will be required to store the given partition/direction
 * halo of 'a'
 */
int MGA_partitionHaloNumel(MGArray *a, int partition, int direction, int h)
{
	// Sanity checks!
	if(partition < 0) DROP_MEX_ERROR("MGA_partitionHaloNumel sanity checks: negative partition id!");
	if(a == NULL) DROP_MEX_ERROR("In MGA_partitionHaloNumel sanity checks: crap, a == NULL!");
	if(partition >= a->nGPUs) DROP_MEX_ERROR("In MGA_partitionHaloNumel sanity checks: crap, partition > # GPUs!");
	if(direction < 1) DROP_MEX_ERROR("In MGA_partitionHaloNumel sanity checks: direction < 1. Did you accidently use XYZ==012?");
	if(direction > 3) DROP_MEX_ERROR("In MGA_partitionHaloNumel sanity checks: direction > 3?");
	if(h < 0) DROP_MEX_ERROR("In MGA_partitionHaloNumel sanity checks: halo size h < 0?");

	int sub[6];
	calcPartitionExtent(a, partition, &sub[0]);

	int haloTransverse = a->partNumel[partition] / sub[2+direction];
	int haloNumel = haloTransverse * h;

	return haloNumel;
}


/* Fetches the indicated face of a partition's cube to a linear swatch of memory,
 * suitable for memcpy or MPI internode halo exchange
 */
int MGA_partitionHaloToLinear(MGArray *a, int partition, int direction, int right, int toHalo, int h, double **linear)
{
	int returnCode = SUCCESSFUL;
	hipSetDevice(a->deviceID[partition]);
	CHECK_CUDA_ERROR("hipSetDevice");

	int sub[6];
	calcPartitionExtent(a, partition, &sub[0]);

	int haloNumel = MGA_partitionHaloNumel(a, partition, direction, h);
	int haloTransverse = haloNumel / h;


	if(linear[0] == NULL) {
//	printf("pointer was null: allocating at cudaCommon.cu:%i\n", __LINE__);
		hipMalloc((void **)linear, 2*haloNumel*sizeof(double));
		returnCode = CHECK_CUDA_ERROR((const char *)"hipMalloc()");
		if(returnCode != SUCCESSFUL) return CHECK_IMOGEN_ERROR(returnCode);
	}

	dim3 blocksize, gridsize;

	switch(direction) {
	case 1: {
		blocksize.x = a->haloSize;
		blocksize.y = SYNCBLOCK;
		blocksize.z = (sub[5] > 1) ? 8 : 1;

		gridsize.x  = ROUNDUPTO(a->dim[1], SYNCBLOCK)/SYNCBLOCK;
		gridsize.y  = 1; gridsize.z = 1;

/*		printf("cuda halo X operation. partition=%i, direction=%i, LR = %i, toHalo = %i, h = %i\n", partition, direction, right, toHalo, h);
		printf("block size: %i %i %i\n", blocksize.x, blocksize.y, blocksize.z);
		printf("grid size: %i %i %i\n", gridsize.x, gridsize.y, gridsize.z);

		printf("partition dimensions being acted on: %i %i %i\n", sub[3], sub[4], sub[5]);
		printf("info about array: ===========================\n");
MGA_debugPrintAboutArray(a);
Leaving this here... there was a strange bug where the wrong h value (hardcoded 3, not h) got passed
here. Somewhere in halo allocation and write, things got turned around and an invalid write occurred.

This invalid write would not trigger a WARP_INVALID_ADDRESS in the debugger, bizzarely enough.
*/
		switch(right + 2*toHalo) {
		/* left read */
		case 0: cudaMGA_haloXrw<0><<<gridsize, blocksize>>>(a->devicePtr[partition] , *linear, sub[3], sub[4], sub[5], h); break;
		/* right read */
		case 1: cudaMGA_haloXrw<1><<<gridsize, blocksize>>>(a->devicePtr[partition] , *linear, sub[3], sub[4], sub[5], h); break;
		/* left write */
		case 2: cudaMGA_haloXrw<2><<<gridsize, blocksize>>>(a->devicePtr[partition] , *linear, sub[3], sub[4], sub[5], h); break;
		/* right write */
		case 3: cudaMGA_haloXrw<3><<<gridsize, blocksize>>>(a->devicePtr[partition] , *linear, sub[3], sub[4], sub[5], h); break;
		default: returnCode = ERROR_CRASH;
		}
		if(returnCode == ERROR_CRASH) break;
		returnCode = CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, a, right + 2*toHalo, (const char *)"cudaMGA_haloXrw");
		break;
	}
	case 2: {
		blocksize.x = blocksize.y = SYNCBLOCK;
		blocksize.z = 1;
		gridsize.x  = a->dim[0]/SYNCBLOCK; gridsize.x += (gridsize.x*SYNCBLOCK < a->dim[0]);
		gridsize.y  = a->dim[2]/SYNCBLOCK; gridsize.y += (gridsize.y*SYNCBLOCK < a->dim[2]);
		switch(right + 2*toHalo) {
		case 0: cudaMGA_haloYrw<0><<<gridsize, blocksize>>>(a->devicePtr[partition] , *linear, sub[3], sub[4], sub[5], h); break;
		case 1: cudaMGA_haloYrw<1><<<gridsize, blocksize>>>(a->devicePtr[partition] , *linear, sub[3], sub[4], sub[5], h); break;
		case 2: cudaMGA_haloYrw<2><<<gridsize, blocksize>>>(a->devicePtr[partition] , *linear, sub[3], sub[4], sub[5], h); break;
		case 3: cudaMGA_haloYrw<3><<<gridsize, blocksize>>>(a->devicePtr[partition] , *linear, sub[3], sub[4], sub[5], h); break;
		default: returnCode = ERROR_CRASH;
		}
		if(returnCode == ERROR_CRASH) break;
		returnCode = CHECK_CUDA_LAUNCH_ERROR(gridsize, blocksize, a, right + 2*toHalo, (const char *)"cudaMGA_haloYrw");
		break;
	}

	case 3: {
		switch(right + 2*toHalo) {
		case 0: hipMemcpy((void *)linear[0], (void *)(a->devicePtr[partition] + haloNumel),                   haloNumel*sizeof(double), hipMemcpyDeviceToDevice); break;
		case 1: hipMemcpy((void *)linear[0], (void *)(a->devicePtr[partition] + (sub[5]-2*h)*haloTransverse), haloNumel*sizeof(double), hipMemcpyDeviceToDevice); break;
		case 2: hipMemcpy((void *)linear[0], (void *)(a->devicePtr[partition]),                               haloNumel*sizeof(double), hipMemcpyDeviceToDevice); break;
		case 3: hipMemcpy((void *)linear[0], (void *)(a->devicePtr[partition] + (sub[5]-h)*haloTransverse),   haloNumel*sizeof(double), hipMemcpyDeviceToDevice); break;
		default: returnCode = ERROR_CRASH;
		}
		if(returnCode == ERROR_CRASH) break;
		returnCode = CHECK_CUDA_ERROR("cudamemcpy");
		break;
	}

	}

return CHECK_IMOGEN_ERROR(returnCode);
}

/* expect invocation with [4*roundup(h/4) BLKy A] threads and [ny/BLKy B 1].rp blocks with "arbitrary" A and B
 * given thread index t.[xyz] block index b.[xyz] and grid size g.[xyz], then consider:
 * x0 = nxL - 2*h + t.x; x1 = t.x;
 * y0 = t.y + BLKy*b.y; z0 = t.z + A*b.y
 * copy from L[x0 + nxL*(y0 + ny*z0)] to R[x1 + nxR*(y0 + ny*z0)]
 * copy from R[x1 + h + nxR*(y0 + ny*z0)] to L[x0 + h + nxL*(y0 + ny*z0)]
 *
 * Extract common subfactors: jump L < L + nxL*(y0 + ny*z0) + x0, R < R + nxR*(y0 + ny*z0) + x1,
 * check y0 < ny, then equations simplify to
 * iterate (k = z0; k < nz; k+=blockIdx.z*blockDim.z)
 *    copy from L[0] to R[0]
 *    copy from R[h] to L[h]
 *    L += nxL*ny*g.y; R =+ nxR*ny*g.y;

 */
__global__ void cudaMGHaloSyncX_p2p(double *L, double *R, int nxL, int nxR, int ny, int nz, int h)
{
	int y0 = threadIdx.y + blockDim.y*blockIdx.x;
	if(y0 >= ny) return;
	int z0 = threadIdx.z + blockDim.z*blockIdx.y;

	/* This will generate unaligned addresses, yes I'm sorry, DEAL WITH IT */
	L += nxL*(y0 + ny*z0) + nxL - 2*h + threadIdx.x;
	R += nxR*(y0 + ny*z0) + threadIdx.x;

	int k;
	int hz = blockDim.z*gridDim.y;
	for(k = z0; k < nz; k+= hz) { /* This implicitly contains: if(z0 >= nz) { return; } */
		// read enough data, for sure
		R[0] = L[0];
		L[h] = R[h];

		L   += nxL*ny*hz;
		R   += nxR*ny*hz;
	}

}

// FIXME: And this ny on both sides, also goddamnit.
/* Expect invocation with [BLKx BLKz 1] threads and [nx/BLKx nz/BLKz 1].rp blocks */
__global__ void cudaMGHaloSyncY_p2p(double *L, double *R, int nx, int nyL, int nyR, int nz, int h)
{
	int x0 = threadIdx.x + blockIdx.x*blockDim.x;
	int z0 = threadIdx.y + blockIdx.y*blockDim.y;

	if((x0 >= nx) || (z0 >= nz)) return;

	L += (x0 + nx*(nyL-2*h + nyL*z0)); // To the plus y extent
	R += (x0 + nx*nyR*z0);        // to the minus y extent

	int i;
	for(i = 0; i < h; i++) {
		L[(i+h)*nx]     = R[(i+h)*nx];
		R[i*nx] = L[i*nx];
	}

}

/* bit 0 = 0: left; bit 0 = 1: right
 * bit 1 = 0: read; bit 1 = 1: write to phi's halo
 */
template<int lr_rw>
__global__ void cudaMGA_haloXrw(double *phi, double *linear, int nx, int ny, int nz, int h)
{
	int y0 = threadIdx.y + blockDim.y*blockIdx.x;
	if(y0 >= ny) return;
	int z0 = threadIdx.z + blockDim.z*blockIdx.y;

	phi += nx*(y0 + ny*z0) + threadIdx.x;
	linear += threadIdx.x + h*(y0+ny*z0);

	switch(lr_rw) {
	case 0: /* left read   */ phi += h; break;
	case 1: /* right read  */ phi += nx - 2*h; break;
	case 2: /* left write  */ break;
	case 3: /* right write */ phi += nx - h; break;
	}

	int k;
	int hz = blockDim.z*gridDim.y;
	for(k = z0; k < nz; k+= hz) { /* This implicitly contains: if(z0 >= nz) { return; } */
		if(lr_rw & 2) {
			phi[0] = linear[0];
		} else {
			linear[0] = phi[0];
		}

		phi    += nx*ny*hz;
		linear += h*ny*hz;
	}

}

/* bit 0 = 0: left; bit 0 = 1: right
 * bit 1 = 0: read; bit 1 = 1: write to phi's halo
 */
template<int lr_rw>
__global__ void cudaMGA_haloYrw(double *phi, double *linear, int nx, int ny, int nz, int h)
{
	int x0 = threadIdx.x + blockIdx.x*blockDim.x;
	int z0 = threadIdx.y + blockIdx.y*blockDim.y;

	// Read extent from array of size {nx ny nz} into array of size {nx h nz}

	if((x0 >= nx) || (z0 >= nz)) return;

	phi    += x0 + nx*ny*z0;
	linear += x0 + nx*h*z0;

	switch(lr_rw) {
	case 0: /* left read   */ phi += nx*h; break;
	case 1: /* right read  */ phi += nx*(ny - 2*h); break;
	case 2: /* left write  */ break;
	case 3: /* right write */ phi += nx*(ny - h); break;
	}


	int i;
	for(i = 0; i < h; i++) {
		if(lr_rw & 2) {
			phi[0] = linear[0];
		} else {
			linear[0] = phi[0];
		}
		phi += nx;
		linear += nx;
	}

}

/* Copes the MGArray pointed to by g into the space at p[0].
 * If partitionFrom is nonnegative, fetches only that partition.
 * If partitionFrom is negative, returns the entire array.
 *
 * If p[0] == NULL, attempts to allocate sufficient storage, which is one of
 *     g->partNumel[partitionFrom] or g->numel.
 * If p[0] != NULL, p[0] must point to sufficient storage. */
// FIXME: this entire crapshow should just make a few calls to hipMemcpy2D/3D
int MGA_downloadArrayToCPU(MGArray *g, double **p, int partitionFrom)
{
	int returnCode = SUCCESSFUL;

	int sub[6];
	long numelOut;
	if(partitionFrom >= 0) {
		numelOut = g->partNumel[partitionFrom];
	} else {
		numelOut = g->numel;
	}

	// Create output numeric array if passed NULL
	// If e.g. returning to MATLAB, it will have already been allocated for us.
	if(p[0] == NULL) {
		*p = (double *)malloc(numelOut * sizeof(double));
	}

	if(p[0] == NULL) {
		PRINT_FAULT_HEADER;
		printf("Host data pointer is null!\nFailed to allocate host storage!\n");
		PRINT_FAULT_FOOTER;
		return ERROR_NULL_POINTER;
	}

	int u, v, w, i;
	int64_t iT, iS;
	double *gmem[g->nGPUs];

	int fromPart, toPart;

	if(partitionFrom >= 0) { // we will fetch only this partition
		fromPart = partitionFrom;
		toPart = partitionFrom + 1;
	} else { // we will fetch all partitions
		fromPart = 0;
		toPart = g->nGPUs;
	}

	for(i = fromPart; i < toPart; i++) {
		gmem[i] = (double *)malloc(g->partNumel[i]*sizeof(double));
		if(gmem[i] == NULL) {
			PRINT_FAULT_HEADER;
			printf("FATAL: Unable to allocate download buffer for GPU array!\n");
			PRINT_FAULT_FOOTER;
			return ERROR_NULL_POINTER;
		}

		hipSetDevice(g->deviceID[i]);

		hipError_t fail = hipMemcpyAsync((void *)gmem[i], (void *)g->devicePtr[i], g->partNumel[i]*sizeof(double), hipMemcpyDeviceToHost);
		returnCode = CHECK_CUDA_ERROR("MGArray_downloadArrayToCPU");
		if(returnCode != SUCCESSFUL) break;
	}
	if(returnCode != SUCCESSFUL) return CHECK_IMOGEN_ERROR(returnCode);

	double *out = p[0];

	int3 ptSize, ptOffset, ptExtent, outOffset, outDims;

	if(partitionFrom >= 0) { // Specific partition: Out dim = that partition
		calcPartitionExtent(g, partitionFrom, &sub[0]);
		outDims.x = sub[3]; outDims.y = sub[4]; outDims.z = sub[5];

	} else {
		outDims.x = g->dim[0];
		outDims.y = g->dim[1];
		outDims.z = g->dim[2];
	}

	double *currentTarget;
	for(i = fromPart; i < toPart; i++) {
		calcPartitionExtent(g, i, &sub[0]);

		ptOffset.x  = sub[0]; ptOffset.y  = sub[1]; ptOffset.z  = sub[2];
		ptSize.x = sub[3]; ptSize.y = sub[4]; ptSize.z = sub[5];

		outOffset = ptOffset;
		ptExtent = ptSize;

		ptOffset.x = 0; ptOffset.y = 0; ptOffset.z = 0;

		currentTarget = gmem[i];

		if(g->nGPUs > 1) {
			// left halo removal
			if((g->addExteriorHalo != 0) || (i > 0)) {
				switch(g->partitionDir) {
				case PARTITION_X: ptExtent.x -= g->haloSize; outOffset.x += g->haloSize; ptOffset.x += g->haloSize; break;
				case PARTITION_Y: ptExtent.y -= g->haloSize; outOffset.y += g->haloSize; ptOffset.y += g->haloSize; break;
				case PARTITION_Z: ptExtent.z -= g->haloSize; outOffset.z += g->haloSize; ptOffset.z += g->haloSize; break;
				}
			}
			// right halo removal
			if((g->addExteriorHalo != 0) || (i < (g->nGPUs-1)))
				switch(g->partitionDir) {
				case PARTITION_X: ptExtent.x -= g->haloSize; break;
				case PARTITION_Y: ptExtent.y -= g->haloSize; break;
				case PARTITION_Z: ptExtent.z -= g->haloSize; break;
				}
		}

		// If we're fetching only 1 partition zap the offset
		if(partitionFrom >= 0) { outOffset.x = outOffset.y = outOffset.z = 0; }

		hipSetDevice(g->deviceID[i]);
		hipDeviceSynchronize();

		for(w = 0; w < ptExtent.z; w++) {
			for(v = 0; v < ptExtent.y; v++) {
				for(u = 0; u < ptExtent.x; u++) {
					iT = u + ptOffset.x + ptSize.x*(v + ptOffset.y + ptSize.y * (w + ptOffset.z));
					iS = u + outOffset.x + outDims.x*(v + outOffset.y + outDims.y * (w + outOffset.z));
					out[iS] = currentTarget[iT];
				}
			}
		}

		free(gmem[i]);
	}


	return SUCCESSFUL;
}

/* Given a pointer to the Matlab array m, checks that g points to an
 * MGArray of the same size as m:
 * if partitionTo is nonnegative, that that partition's extent equals
 * the size of m
 * if partitionTo is negative, that the g->dim equals the size of m.
 * and then initiates the transfer.
 */
int MGA_uploadMatlabArrayToGPU(const mxArray *m, MGArray *g, int partitionTo)
{

if(m == NULL) return -1;
if(g == NULL) return -1;

mwSize ndims = mxGetNumberOfDimensions(m);
if(ndims > 3) { DROP_MEX_ERROR((const char *)"Input array has more than 3 dimensions!"); }

const mwSize *arraydims = mxGetDimensions(m);

int j;
int failed = 0;

for(j = 0; j < ndims; j++) { 
	if(arraydims[j] != g->dim[j]) failed = 1;
}

if(failed) {
	PRINT_FAULT_HEADER;
	printf("Matlab array was %i dimensional, dims [", (int)ndims);
	for(j = 0; j < ndims; j++) { printf("%i ", (int)arraydims[j]); }
	printf("].\nGPU_Type target array was of size [%i %i %i] which is not the same. Not happy :(.\n", g->dim[0], g->dim[1], g->dim[2]);
	PRINT_FAULT_FOOTER;
	return ERROR_INVALID_ARGS;
	}

return CHECK_IMOGEN_ERROR(MGA_uploadArrayToGPU(mxGetPr(m), g, partitionTo));

}

/* Assuming that srcPtr and dstPtr are the pointers returned by hipMalloc()s to arrays of size
 * sizeof(src) = sDim and sizeof(dst) = dDim, and we wish to copy a block of size imax from
 * src[sOffset] ... src[sOffset + imax] to dst[dOffset] ... dst[dOffset + imax],
 * the host code must pass this kernel
 * src = srcPtr + sOffset.x + sDim.x*(sOffset.y+sDim.y*sOffset.z)
 * dst = dstPtr + dOffset.x + dDim.x*(dOffset.y+dDim.z*dOffset.z)
 * i.e. perform the base offset translate itself, then this kernel will copy the data from
 * src to dst.
 * 
 * This kernel may be launched with arbitrary x+y block size and dimensions and z size of one. */
__global__ void cukern_BlockSubsetCopy(double *dst, double *src, int3 dDim, int3 sDim, int3 imax)
{
int myx = threadIdx.x + blockIdx.x * blockDim.x;
int myy = threadIdx.y + blockIdx.y * blockDim.y;

if((myx >= imax.x) || (myy >= imax.y)) return;

dst += myx + dDim.x * myy;
src += myx + sDim.x * myy;
// Now done with those variables, compiler can dump them

int d_nxy = dDim.x*dDim.y;
int s_nxy = sDim.x*sDim.y;

int q;
for(q = 0; q < imax.z; q++) {
	dst[0] = src[0];
	dst += d_nxy;
	src += s_nxy;
}

}

__global__ void cukern_BlockSubsetCopyLarge(double *dst, double *src, int3 dDim, int3 sDim, int3 imax)
{
int myx = threadIdx.x + blockIdx.x * blockDim.x;
int myy = threadIdx.y + blockIdx.y * blockDim.y;

int d_nxy = dDim.x*dDim.y;
int s_nxy = sDim.x*sDim.y;

int q, u, v;
for(u = myx; u < imax.x; u+= blockDim.x*gridDim.x) {
	for(v = myy; v < imax.y; v+= blockDim.y*gridDim.y) {
		for(q = 0; q < imax.z; q++) {
			dst[u+dDim.x*(v+d_nxy*q)] = src[u+sDim.x*(v+s_nxy*q)];
		}
	}
}

}

/* Assuming that p points to an array whose size is compatible with either
 * the whole of g (partitionOnto < 0) or a specific partition of g (if
 * partitionOnto >= 0), transfers elements of p to g.
 */
int MGA_uploadArrayToGPU(double *p, MGArray *g, int partitionTo)
{
	int returnCode = SUCCESSFUL;
	int sub[6];

	// Create output numeric array if passed NULL
	// If e.g. returning to MATLAB, it will have already been allocated for us.
	if(p == NULL) {
		PRINT_FAULT_HEADER;
		printf("Host data pointer is null!");
		PRINT_FAULT_FOOTER;
		return ERROR_NULL_POINTER;
	}

	int i;
	double *gmem[g->nGPUs];

	int fromPart, toPart;

	if(partitionTo >= 0) { // Uploading to a single partition
		fromPart = partitionTo;
		toPart = partitionTo + 1;
	} else { // we will fetch all partitions
		fromPart = 0;
		toPart = g->nGPUs;
	}

	/* By way of explanation:
	 * This is done by coping the whole array to every device and then invoking
	 * a subset copy function on the GPU because to achieve any level of acceptable
	 * performance on the host would require OMP #pragmas on the for() for() for() loop
 	 * that copies the subset on the host, but getting compiler flags to work with
	 * just matlab and cuda proved difficult enough */

	for(i = fromPart; i < toPart; i++) {
		long NE;
		if(g->haloSize==0) { // partition cloned: g->dim will contain a dim Xed by nGPUs somewhere
			NE = g->partNumel[i];
		} else { // normal: g->dim equals Matlab array size
			NE = g->dim[0] * g->dim[1] * g->dim[2];
		}

		hipSetDevice(g->deviceID[i]);
		CHECK_CUDA_ERROR("hipSetDevice");
		hipMalloc((void **)(&gmem[i]), NE * sizeof(double));
		CHECK_CUDA_ERROR("hipMalloc");
		hipMemcpyAsync((void *)gmem[i], (void *)p, NE * sizeof(double), hipMemcpyHostToDevice);

		if(gmem[i] == NULL) {
			PRINT_FAULT_HEADER;
			printf("Unable to allocate upload buffer!\n");
			PRINT_FAULT_FOOTER;
			return ERROR_NULL_POINTER;

		}
	}

	int3 ptSize, ptOff, partExtent, readOff;
	int *usedims;

	for(i = fromPart; i < toPart; i++) {
		calcPartitionExtent(g, i, &sub[0]);

		ptOff.x  = sub[0]; ptOff.y  = sub[1]; ptOff.z  = sub[2];
		ptSize.x = sub[3]; ptSize.y = sub[4]; ptSize.z = sub[5];

		readOff = ptOff;
		partExtent = ptSize;

		ptOff.x = 0; ptOff.y = 0; ptOff.z = 0;

		if(g->nGPUs > 1) {
			// left halo removal
			if((g->addExteriorHalo != 0) || (i > 0)) {
				switch(g->partitionDir) {
				case PARTITION_X: partExtent.x -= g->haloSize; readOff.x += g->haloSize; ptOff.x += g->haloSize; break;
				case PARTITION_Y: partExtent.y -= g->haloSize; readOff.y += g->haloSize; ptOff.y += g->haloSize; break;
				case PARTITION_Z: partExtent.z -= g->haloSize; readOff.z += g->haloSize; ptOff.z += g->haloSize; break;
				}
			}
			// right halo removal
			if((g->addExteriorHalo != 0) || (i < (g->nGPUs-1)))
				switch(g->partitionDir) {
				case PARTITION_X: partExtent.x -= g->haloSize; break;
				case PARTITION_Y: partExtent.y -= g->haloSize; break;
				case PARTITION_Z: partExtent.z -= g->haloSize; break;
				}
		}

		// If we're fetching only 1 partition: zap the offset
		if(partitionTo >= 0) {
			readOff.x = readOff.y = readOff.z = 0;
			usedims = &sub[3];
		} else {
			usedims = &g->dim[0];
		}

		hipSetDevice(g->deviceID[i]);
		CHECK_CUDA_ERROR("hipSetDevice");

		dim3 blockdim = makeDim3(16, 16, 1);
		// partially ameliorate the problem if it occurs:
		if(partExtent.x*partExtent.y >= 256) {
			if(partExtent.x < 16) {
				// we want to maintain proper striding so keep x a power of 2
				blockdim.x = 8;
				while(blockdim.x*blockdim.y < 256) blockdim.y *= 2;
			}
			if(partExtent.y < 16) {
				blockdim.y = partExtent.y;
				while(blockdim.x * blockdim.y < 256) blockdim.x *= 2;
		}
		}

		// block size should result in reasonable dims extent for large vector-like objects
		dim3 griddim = makeDim3(
				ROUNDUPTO(partExtent.x, blockdim.x)/blockdim.x,
				ROUNDUPTO(partExtent.y, blockdim.y)/blockdim.y,
				1);

		// or it may not!
		int useLargeCopy = 0;

		if(griddim.x > 512) { griddim.x = 512; useLargeCopy = 1; }
		if(griddim.y > 512) { griddim.y = 512; useLargeCopy = 1; }
		
		// Potential trip-up: extremely long, narrow array (limiting case: a vector)

		double *dstos = g->devicePtr[i] + ptOff.x + ptSize.x * (ptOff.y + ptSize.y * ptOff.z);
		double *srcos = gmem[i] + readOff.x + usedims[0] * (readOff.y + usedims[1] * readOff.z);
		int3 srcDims = makeInt3(usedims[0], usedims[1], usedims[2]);

		if(useLargeCopy) {
			cukern_BlockSubsetCopyLarge<<<griddim, blockdim>>>(dstos, srcos, ptSize, srcDims, partExtent);
		} else {
			cukern_BlockSubsetCopy<<<griddim, blockdim>>>(dstos, srcos, ptSize, srcDims, partExtent);
		}
		returnCode = CHECK_CUDA_ERROR((const char *)"MGArray_uploadArrayToGPU");
		if(returnCode != SUCCESSFUL) break;

	}

	for(i = fromPart; i < toPart; i++) {
		hipSetDevice(g->deviceID[i]);
		returnCode = CHECK_CUDA_ERROR("hipSetDevice");
		if(returnCode != SUCCESSFUL) break;
		hipDeviceSynchronize();
		returnCode = CHECK_CUDA_ERROR("hipDeviceSynchronize");
		if(returnCode != SUCCESSFUL) break;
		hipFree(gmem[i]);
		returnCode = CHECK_CUDA_ERROR("hipFree");
		if(returnCode != SUCCESSFUL) break;
	}

	if(returnCode != SUCCESSFUL) {
		return CHECK_IMOGEN_ERROR(returnCode);
	}

	returnCode = MGA_exchangeLocalHalos(g, 1);
	return CHECK_IMOGEN_ERROR(returnCode);

}

/* Given a pointer to a FluidManager class, accesses the fluids stored
 * within it. */
int MGA_accessFluidCanister(const mxArray *canister, int fluidIdx, MGArray *fluid)
{
	/* Access the FluidManager canisters */
	mxArray *fluidPtrs[3];
	fluidPtrs[0] = mxGetProperty(canister, fluidIdx,(const char *)("mass"));
	if(fluidPtrs[0] == NULL) {
		PRINT_FAULT_HEADER;
		printf("Unable to fetch 'mass' property from canister\nNot a FluidManager class?\n");
		PRINT_FAULT_FOOTER;
		return ERROR_INVALID_ARGS;
	}
	fluidPtrs[1] = mxGetProperty(canister, fluidIdx,(const char *)("ener"));
	if(fluidPtrs[1] == NULL) {
		PRINT_FAULT_HEADER;
		printf("Unable to fetch 'ener' property from canister\nNot a FluidManager class?\n");
		PRINT_FAULT_FOOTER;
		return ERROR_INVALID_ARGS;
	}
	fluidPtrs[2] = mxGetProperty(canister, fluidIdx,(const char *)("mom"));
	if(fluidPtrs[2] == NULL) {
		PRINT_FAULT_HEADER;
		printf("Unable to fetch 'mom' property from canister\nNot a FluidManager class?\n");
		PRINT_FAULT_FOOTER;
		return ERROR_INVALID_ARGS;
	}

	int status = MGA_accessMatlabArrays((const mxArray **)&fluidPtrs[0], 0, 1, &fluid[0]);
	if(status != SUCCESSFUL) return CHECK_IMOGEN_ERROR(status);
    status = MGA_accessMatlabArrayVector(fluidPtrs[2], 0, 2, &fluid[2]);
    if(status != SUCCESSFUL) return CHECK_IMOGEN_ERROR(status);

    return SUCCESSFUL;
}

ThermoDetails accessMatlabThermoDetails(const mxArray *thermstruct)
{
	ThermoDetails thermo;
	thermo.gamma = derefXdotAdotB_scalar(thermstruct, "gamma", NULL);

	thermo.m     = derefXdotAdotB_scalar(thermstruct, "mass", NULL);

	thermo.mu0   = derefXdotAdotB_scalar(thermstruct, "dynViscosity", NULL);
	thermo.muTindex = derefXdotAdotB_scalar(thermstruct, "viscTindex", NULL);
	thermo.sigma0 = derefXdotAdotB_scalar(thermstruct, "sigma", NULL);
	thermo.sigmaTindex= derefXdotAdotB_scalar(thermstruct, "sigmaTindex", NULL);

	thermo.kBolt = derefXdotAdotB_scalar(thermstruct, "kBolt", NULL);
	thermo.Cisothermal = derefXdotAdotB_scalar(thermstruct, "Cisothermal", NULL);

	return thermo;
}

GeometryParams accessMatlabGeometryClass(const mxArray *geoclass)
{
	GeometryParams g;
	double v[3];

	g.Rinner = derefXdotAdotB_scalar(geoclass, "pInnerRadius", NULL);
	derefXdotAdotB_vector(geoclass, "d3h", NULL, &g.h[0], 3);

	derefXdotAdotB_vector(geoclass, "frameRotationCenter", NULL, &g.frameRotateCenter[0], 3);
	g.frameOmega = derefXdotAdotB_scalar(geoclass, "frameRotationOmega", NULL);

	int shapenum = derefXdotAdotB_scalar(geoclass, "pGeometryType", NULL);

	switch(shapenum) {
	case 1: g.shape = SQUARE; break;
	case 2: g.shape = CYLINDRICAL; break;
	// default: ?
	}

	derefXdotAdotB_vector(geoclass, "affine", NULL, &v[0], 3);
	g.x0 = v[0];
	g.y0 = v[1];
	g.z0 = v[2];

	return g;
}

/* A utility to ease access to Matlab structures/classes: fetches in(idx).{fieldA}.{fieldB}
 * or in(idx).{fieldA} if fieldB is NULL and returns the resulting mxArray* */
mxArray *derefXatNdotAdotB(const mxArray *in, int idx, const char *fieldA, const char *fieldB)
{

	if(fieldA == NULL) mexErrMsgTxt("In derefAdotBdotC: fieldA null!");

	mxArray *A; mxArray *B;
	mxClassID t0 = mxGetClassID(in);

	int snum = strlen("Failed to read field fieldA in X.A.B") + (fieldA != NULL ? strlen(fieldA) : 5) + (fieldB != NULL ? strlen(fieldB) : 5) + 10;
	char *estring;

	if(t0 == mxSTRUCT_CLASS) { // Get structure field from A
		A = mxGetField(in, idx, fieldA);

		if(A == NULL) {
			estring = (char *)calloc(snum, sizeof(char));
			sprintf(estring,"Failed to get X.%s", fieldA);
			mexErrMsgTxt(estring);
		}
	} else { // Get field struct A and fail if it doesn't work
		A = mxGetProperty(in, idx, fieldA);

		if(A == NULL) {
			estring = (char *)calloc(snum, sizeof(char));
			sprintf(estring,"Failed to get X.%s", fieldA);
			mexErrMsgTxt(estring);
		}
	}

	if(fieldB != NULL) {
		t0 = mxGetClassID(A);
		if(t0 == mxSTRUCT_CLASS) {
			B = mxGetField(A, idx, fieldB);
		} else {
			B = mxGetProperty(A, idx, fieldB);
		}

		if(B == NULL) {
			estring = (char *)calloc(snum, sizeof(char));
			sprintf(estring,"Failed to get X.%s.%s", fieldA, fieldB);
			mexErrMsgTxt(estring);
		}

		return B;
	} else {
		return A;
	}
}

/* A utility to ease access to Matlab structures/classes: fetches in.{fieldA}.{fieldB}
 * or in.{fieldA} if fieldB is NULL and returns the resulting mxArray* */
mxArray *derefXdotAdotB(const mxArray *in, const char *fieldA, const char *fieldB)
{
	return derefXatNdotAdotB(in, 0, fieldA, fieldB);
}

/* Fetches in.{fieldA}.{fieldB}, or in.{fieldA} if fieldB is NULL,
 * and returns the first double element of this.
 */
double derefXdotAdotB_scalar(const mxArray *in, const char *fieldA, const char *fieldB)
{
	mxArray *u = derefXdotAdotB(in, fieldA, fieldB);

	if(u != NULL) return *mxGetPr(u);

	return NAN;
}

/* Fetches in.{fieldA}.{fieldB}, or in.{fieldA} if fieldB is NULL,
 * and copies the first N elements of this into x[0, ..., N-1] if we get
 * a valid double *, or writes NANs if we do not.
 * If the Matlab array has fewer than N elements, truncates the copy.
 */
void derefXdotAdotB_vector(const mxArray *in, const char *fieldA, const char *fieldB, double *x, int N)
{
	mxArray *u = derefXdotAdotB(in, fieldA, fieldB);

	int Nmax = mxGetNumberOfElements(u);
	N = (N > Nmax) ? Nmax : N;

	double *d = mxGetPr(u);
	int i;

	if(d != NULL) {
		for(i = 0; i < N; i++) { x[i] = d[i]; } // Give it the d.
	} else {
		for(i = 0; i < N; i++) { x[i] = NAN; }
	}

}

void getTiledLaunchDims(int *dims, dim3 *tileDim, dim3 *halo, dim3 *blockdim, dim3 *griddim)
{
	blockdim->x = tileDim->x + halo->x;
	blockdim->y = tileDim->y + halo->y;
	blockdim->z = tileDim->z + halo->z;

	griddim->x = dims[0] / tileDim->x; griddim->x += ((griddim->x * tileDim->x) < dims[0]);
	griddim->y = dims[1] / tileDim->y; griddim->y += ((griddim->y * tileDim->y) < dims[1]);
	griddim->z = dims[2] / tileDim->z; griddim->z += ((griddim->z * tileDim->z) < dims[2]);
}

/* This should be checked after every GPU kernel launch: It provides detailed metadata feedback to
 * greatly facilitate debugging. If it does not return SUCCESSFUL, the function should
 * abort and return its return value: This effectively prints a compiled code backtrace.
 */
int checkCudaLaunchError(hipError_t E, dim3 blockdim, dim3 griddim, MGArray *a, int i, const char *srcname, const char *fname, int lname)
{
	if(E == hipSuccess) return SUCCESSFUL;

	int myrank;
	MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
	printf("========== FAULT FROM CUDA API (%s:%i), RANK %i\n", fname, lname, myrank);

	printf("Caught CUDA error %s -> %s\n", errorName(E), hipGetErrorString(E));
	printf("Code's description of what it just did: %s\n", srcname);
	printf("Rx'd the integer: %i\n", i);

	if(a == NULL) {
		PRINT_FAULT_HEADER;
		printf("CUDA reported a problem after kernel launch.\nBut no MGArray passed to error checker... ?!?!?!?\nReturning crash condition...\n");
		PRINT_FAULT_FOOTER;
		return ERROR_CRASH;
	}

	MGA_debugPrintAboutArray(a);
	printf("Block and grid dims of kernel: <%i %i %i>, <%i %i %i>\n", blockdim.x, blockdim.y, blockdim.z, griddim.x, griddim.y, griddim.z);

    PRINT_FAULT_FOOTER;

	return ERROR_CUDA_BLEW_UP;
}

/* Emit a verbose description of the MGArray living at *x. */
void MGA_debugPrintAboutArray(MGArray *x)
{
	int n = x->nGPUs;
	int rank;
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	printf("========== RANK %i DEBUG INFORMATION ABOUT ARRAY\n", rank);
	printf("  This array's address: %lx\n", (unsigned long)x);
	printf("===== Device information\n");
	printf("Array distributed onto    %i GPUs\n", x->nGPUs);
	int j;
	for(j = 0; j < n; j++) { printf("Device %i: [%i | %lx]\n", j, x->deviceID[j], (unsigned long)x->devicePtr[j]); }

	printf("Array's host-side extent: [%i %i %i]\n", x->dim[0], x->dim[1], x->dim[2]);
	printf("Array's host #elements  : %li; ", x->numel);
	if(x->numSlabs > 1) {
		printf("Array IS A REAL ALLOCATION with %i slabs.\n", x->numSlabs);
	} else {
		printf("Array IS A SLAB REFERENCE  index number %i\n", -x->numSlabs);
	}
	printf("Array's slab pitches    : [");
	for(j = 0; j < n; j++) { printf("%li ", x->slabPitch[j]); }
	printf("]\n");
	printf("Partition halo size     : %i\n", x->haloSize);
	printf("Partition direction     : %i\n", x->partitionDir);
	printf("Halo added to exterior? : %c\n", x->addExteriorHalo ? 'y' : 'n');
	printf("Permutation tag value   : %i\n", x->permtag);
	printf("Which represents        : [%i %i %i] stride ordering\n", x->currentPermutation[0], x->currentPermutation[1], x->currentPermutation[2]);
	printf("circularBoundaryBits    : %i\n", x->circularBoundaryBits);
	printf("Matlab source class idx : %i\n", x->mlClassHandleIndex);
	printf("==========\n");

}

void MGA_debugPrintAboutArrayBrief(MGArray *x)
{
        int n = x->nGPUs;
        int rank;
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        printf("RANK %i BRIEF DEBUG INFORMATION ABOUT ARRAY: Distributed onto %i GPUs. Pointers: ", rank, x->nGPUs);
        int j;
        for(j = 0; j < n; j++) { printf("%lx ", (unsigned long)x->devicePtr[j]); }
        printf("RANK %i Array's host-side extent = [%i %i %i]; ", rank, x->dim[0], x->dim[1], x->dim[2]);
        printf("RANK %i Halo size = %i; partition dir = %i, exteriorHalo=%i, boundary bits=%i\n", rank, x->haloSize, x->partitionDir, x->addExteriorHalo, x->circularBoundaryBits);
}

/* This should be polled after CUDA API calls. In the event of a problem, it provides detailed
 * metadata feedback to assist with debugging. If it returns unsuccessful, the function
 * should immediately cleanup & return its return value.
 */
int checkCudaError(const char *where, const char *fname, int lname)
{
	hipError_t epicFail = hipGetLastError();
	if(epicFail == hipSuccess) return SUCCESSFUL;

	int myrank;
	MPI_Comm_rank(MPI_COMM_WORLD, &myrank);

	printf("Rank %i | In (%s:%i): === FAULT FROM CUDA API ===\n", myrank, fname, lname, myrank);
	printf("\tcudaCheckError reported %s: %s\n", errorName(epicFail), hipGetErrorString(epicFail));

	return ERROR_CUDA_BLEW_UP;
}

/* This function facilitates error feedback that assists with debugging.
 * Functions which can fail should return an error integer.
 * Every call to a function which can fail should use the CHECK_IMOGEN_ERROR() macro
 * and abort in the event it returns other than SUCCESSFUL;
 * This process will print a backtrace of where the error occurred, leading back
 * to the invoking mexFunction entry point.
 *
 * The mexFunction itself, upon detecting failure, should cause a mexError, which will
 * cause the invocation of a similar backtrace in the Matlab layer
 *
 * NOTE: Don't use this directly, use the CHECK_IMOGEN_ERROR(errorcode) macro which will automatically
 * have the correct filename, function name and line number.
 */
int checkImogenError(int errtype, const char *infile, const char *infunc, int atline)
{
	if(errtype == SUCCESSFUL) return SUCCESSFUL;
	int mpirank;
	MPI_Comm_rank(MPI_COMM_WORLD, &mpirank);

	const char *estring;

	switch(errtype)
	{
	case ERROR_INVALID_ARGS:              estring = "One or more invalid arguments caught."; break;
	case ERROR_CRASH:                     estring = "Evidently we can/will/must crash.\n"; break;
	case ERROR_NULL_POINTER:              estring = "Null pointer caught."; break;
	case ERROR_GET_GPUTAG_FAILED:         estring = "Attempt to get GPU tag failed."; break;
	case ERROR_DESERIALIZE_GPUTAG_FAILED: estring = "Deserialization of gputag -> MGArray failed."; break;
	case ERROR_CUDA_BLEW_UP:              estring = "CUDA API returned an error. Crashing."; break;
	case ERROR_NOIMPLEMENT:               estring = "Required functionality not implemented."; break;
	default:                              estring = "Invalid error code or no error."; break;
	}
	printf("Rank %i | In %s (%s:%i): %s\n", mpirank, infunc, infile, atline, estring);
	return errtype;
}

#define MAX_BAD_VALUES 1024

/* Trawls through the given pointer for the given length
 * Looks for any NAN or INF values; These are characteristic of the code taking an
 * explosive dump. Store the addresses as they are found in evils[].
 * evils[0] counts # of invalid values
 */
__global__ void cukern_dbcheck_array(int *evils, double *array, int Nmax)
{

	int x = threadIdx.x + blockIdx.x*blockDim.x;

	double foo; int w;
	while(x < Nmax) {
		foo = array[x];
		if( (isnan(foo)) || (isinf(foo)) ) {
			w = atomicAdd(evils, 1);
			if(w > (MAX_BAD_VALUES-1)) break;
			evils[w] = x;
		}
		x += blockDim.x*gridDim.x;

	}

}

int dbgfcn_CheckArrayVals(MGArray *x, int crashit) { return dbgfcn_CheckArrayVals(x, 1, crashit); }

/* Calls a kernel to run through all partitions of MGArray x,
 * storing addresses of invalid data in a separate array (up to 1024).
 */
int dbgfcn_CheckArrayVals(MGArray *x, int maxslab, int crashit)
{
int rank;
MPI_Comm_rank(MPI_COMM_WORLD, &rank);

int fail = SUCCESSFUL;

int badnews = 0;

int hostBadVals[MAX_BAD_VALUES];

if(maxslab == 0) { maxslab = 1; }
if(maxslab > x->numSlabs) { maxslab = x->numSlabs; }

int *evilAddresses[x->nGPUs];


int i, j;

for(i = 0; i < x->nGPUs; i++) {
	hipSetDevice(x->deviceID[i]);
	hipMalloc(&evilAddresses[i], MAX_BAD_VALUES*sizeof(int));
}

for(j = 0; j < maxslab; j++) {
	for(i = 0; i < x->nGPUs; i++) {
		hipSetDevice(x->deviceID[i]);

		fail = CHECK_CUDA_ERROR("set device");

		int qq = 1;
		hipMemcpy((void *)evilAddresses[i], (void *)&qq, 1*sizeof(int), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		fail = CHECK_CUDA_ERROR("memcopy");
		cukern_dbcheck_array<<<16, 256>>>(evilAddresses[i], x->devicePtr[i] + (j*x->slabPitch[i]/8), x->partNumel[i]);
		fail = CHECK_CUDA_ERROR("cukern_dbcheck_array");
		hipMemcpy((void *)&qq, (void *)evilAddresses[i], 1*sizeof(int), hipMemcpyDeviceToHost);
		fail = CHECK_CUDA_ERROR("memcopy");
		if(qq > 1) {
			printf("Rank %i reporting: dbgfcn_CheckArrayVals investigated slab %i, %lx[0] to %lx[%i] and hit %i invalid values in partition %i!\n", rank, j, (unsigned long)x->devicePtr[i], (unsigned long)x->devicePtr[i], x->partNumel[i]-1, qq, i);
			printf("If repeatable, set break %s:%i, data has been D/Led to host for convenience.\n", __FILE__, __LINE__);
			printf("Dumping up to the first 32 problems below:\n");
			hipMemcpy((void *)&hostBadVals[1], (void *)evilAddresses[i], (qq-1)*sizeof(int), hipMemcpyDeviceToHost);
			hipDeviceSynchronize();
			fail = CHECK_CUDA_ERROR("memcopy");

			int dispmax = qq; if(qq > 32) { dispmax = 32; }

			int3 pad, gad;
			int ctbad;
			int sub[6];
			calcPartitionExtent(x, j, &sub[0]);

			for(ctbad = 0; ctbad < dispmax; ctbad++) {
				decodePartitionAddress(hostBadVals[ctbad+1], &sub[0], &pad, &gad);
				printf("Rank %i: Bad address #%i=%i, partition idx = [%i %i %i], global idx = [%i %i %i]\n", rank, ctbad, hostBadVals[ctbad+1], pad.x, pad.y, pad.z, gad.x, gad.y, gad.z);
			}
			badnews = 1;
			break;
		}
	}

	if(badnews) {
		printf("Rank %i providing information on array where fault was found:\n", rank);
		MGA_debugPrintAboutArray(x);

		
		if(crashit) {
			printf("Rank %i returning error to cause automatic backtrace output.\n", rank);
			hipFree(evilAddresses);
			return ERROR_CRASH;
		}
	}

}

for(i = 0; i < x->nGPUs; i++) {
	hipSetDevice(x->deviceID[i]);
	hipFree(evilAddresses[i]);
}

return fail;

}

/* Invokes dbgfcn_CheckArrayVals on fluid[0] through fluid[4] */
int dbgfcn_CheckFluidVals(MGArray *fluid, int crashit)
{

	int badnews;
	int i;
	for(i = 0; i < 4; i++) {
		badnews = dbgfcn_CheckArrayVals(fluid + i, 1, crashit);
		if(CHECK_IMOGEN_ERROR(badnews) != SUCCESSFUL) { return badnews; }
	}

	return 0;
}

/* This function should be used the mexFunction entry points (AND NOWHERE ELSE) to signal Matlab of problems.
 * NOTE: Don't call this directly, use DROP_MEX_ERROR("string") to automatically
 * fill in the file and line numbers correctly.
 */
void dropMexError(const char *excuse, const char *infile, int atline)
{
	static char turd[512];
	snprintf(turd, 511, "Bad news bears:\n\t%s\n\tLocation was %s:%i", excuse, infile, atline);
	mexErrMsgTxt(turd);
}

void printdim3(char *name, dim3 dim)
{ printf("dim3 %s is [%i %i %i]\n", name, dim.x, dim.y, dim.z); }

void printgputag(char *name, int64_t *tag)
{ printf("gputag %s is [*=%lu dims=%lu size=(%lu %lu %lu)]\n", name, tag[0], tag[1], tag[2], tag[3], tag[4]); }

/* Accepts an MPI_Op reduction type and returns an enum appropriate
 * for my templated accelerated reduce functions.
 */
MGAReductionOperator MGAReductionOperator_mpi2mga(MPI_Op mo)
{
	MGAReductionOperator op = MGA_OP_SUM;

	if(mo == MPI_SUM) op = MGA_OP_SUM;
	if(mo == MPI_PROD)op = MGA_OP_PROD;
	if(mo == MPI_MAX) op = MGA_OP_MAX;
	if(mo == MPI_MIN) op = MGA_OP_MIN;
	return op;
}
/* Accepts my enumerated reduce operators & converts them to MPI_Op types
 * for global operations.
 */
MPI_Op MGAReductionOperator_mga2mpi(MGAReductionOperator op)
{
	MPI_Op mo = MPI_SUM;

	switch(op) {
	case MGA_OP_SUM: mo = MPI_SUM; break;
	case MGA_OP_PROD:mo = MPI_PROD; break;
	case MGA_OP_MIN: mo = MPI_MIN; break;
	case MGA_OP_MAX: mo = MPI_MAX; break;
	}
return mo;
}


#define NOM(x) if(E == x) { static const char err[]=#x; return err; }

const char *errorName(hipError_t E)
{
	/* Written the stupid way because nvcc is idiotically claims these are all "case inaccessible" if it's done with a switch.

WRONG, asshole! */
	// OM...
	NOM(hipSuccess)
		NOM(hipErrorMissingConfiguration)
		NOM(hipErrorOutOfMemory)
		NOM(hipErrorNotInitialized)
		NOM(hipErrorLaunchFailure)
		NOM(hipErrorPriorLaunchFailure)
		NOM(hipErrorLaunchTimeOut)
		NOM(hipErrorLaunchOutOfResources)
		NOM(hipErrorInvalidDeviceFunction)
		NOM(hipErrorInvalidConfiguration)
		NOM(hipErrorInvalidDevice)
		NOM(hipErrorInvalidValue)
		NOM(hipErrorInvalidPitchValue)
		NOM(hipErrorInvalidSymbol)
		NOM(hipErrorMapFailed)
		NOM(hipErrorUnmapFailed)
		NOM(cudaErrorInvalidHostPointer)
		NOM(hipErrorInvalidDevicePointer)
		NOM(hipErrorInvalidTexture)
		NOM(cudaErrorInvalidTextureBinding)
		NOM(hipErrorInvalidChannelDescriptor)
		NOM(hipErrorInvalidMemcpyDirection)
		NOM(cudaErrorAddressOfConstant)
		NOM(cudaErrorTextureFetchFailed)
		NOM(cudaErrorTextureNotBound)
		NOM(cudaErrorSynchronizationError)
		NOM(cudaErrorInvalidFilterSetting)
		NOM(cudaErrorInvalidNormSetting)
		NOM(cudaErrorMixedDeviceExecution)
		NOM(hipErrorDeinitialized)
		NOM(hipErrorUnknown)
		NOM(cudaErrorNotYetImplemented)
		NOM(cudaErrorMemoryValueTooLarge)
		NOM(hipErrorInvalidHandle)
		NOM(hipErrorNotReady)
		NOM(hipErrorInsufficientDriver)
		NOM(hipErrorSetOnActiveProcess)
		NOM(cudaErrorInvalidSurface)
		NOM(hipErrorNoDevice)
		NOM(hipErrorECCNotCorrectable)
		NOM(hipErrorSharedObjectSymbolNotFound)
		NOM(hipErrorSharedObjectInitFailed)
		NOM(hipErrorUnsupportedLimit)
		NOM(cudaErrorDuplicateVariableName)
		NOM(cudaErrorDuplicateTextureName)
		NOM(cudaErrorDuplicateSurfaceName)
		NOM(cudaErrorDevicesUnavailable)
		NOM(hipErrorInvalidImage)
		NOM(hipErrorNoBinaryForGpu)
		NOM(cudaErrorIncompatibleDriverContext)
		NOM(hipErrorPeerAccessAlreadyEnabled)
		NOM(hipErrorPeerAccessNotEnabled)
		NOM(hipErrorContextAlreadyInUse)
		NOM(hipErrorProfilerDisabled)
		NOM(hipErrorProfilerNotInitialized)
		NOM(hipErrorProfilerAlreadyStarted)
		NOM(hipErrorProfilerAlreadyStopped)
		/*hipErrorAssert
cudaErrorTooManyPeers
hipErrorHostMemoryAlreadyRegistered
hipErrorHostMemoryNotRegistered
hipErrorOperatingSystem*/
		NOM(cudaErrorStartupFailure)
		// ... NOM, ASSHOLE!
		return NULL;
}



