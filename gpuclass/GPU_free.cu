#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"
#include "mpi_common.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	// wrapper for hipFree().
	if((nlhs != 0) || (nrhs == 0)) mexErrMsgTxt((const char *)"GPU_free: syntax is GPU_free(arbitrarily many GPU_Types, gpu tags, or ImogenArrays)");

	int returnCode = CHECK_CUDA_ERROR((const char *)"Entering GPU_free()");
	if(returnCode != SUCCESSFUL)
		return;

	MGArray t[nrhs];

	returnCode = MGA_accessMatlabArrays(prhs, 0, nrhs-1, &t[0]);
	if(returnCode != SUCCESSFUL) {
		CHECK_IMOGEN_ERROR(returnCode);
		return;
	}

	int i;
	for(i = 0; i < nrhs; i++) {
		returnCode = MGA_delete(t+i);
		if(returnCode != SUCCESSFUL) break;
	}

	if(returnCode != SUCCESSFUL) CHECK_IMOGEN_ERROR(returnCode);

	return;
}
