#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

#include "mpi.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "nvToolsExt.h"

#include "cudaCommon.h"
#include "cudaFluidStep.h"

#include "cflTimestep.h"

#include "fluidMethod.h"

/* cflTimestep.cu responds to the following -D options:
 * USE_STATIC_CHALLOC: If defined, allocates a static array of page locked gpu-accessible memory 
 *                     used in max() reductions. This array is grown on demand, but is small.
 * SYNCMEX: If set, calls MGA_sledgehammerSequentialize before exit
 */

/* THIS FUNCTION:
   directionalMaxFinder has three different behaviors depending on how it is called.
   m = directionalMaxFinder(array) will calculate the global maximum of array
   c = directionalMaxFinder(a1, a2, direct) will find the max of |a1(r)+a2(r)| in the
      'direct' direction (1=X, 2=Y, 3=Z)
   c = directionalMaxFinder(rho, c_s, px, py, pz) will specifically calculate the x direction
       CFL limiting speed, max(|px/rho| + c_s)
    */

template <int simulationDimension, geometryType_t shape, FluidMethods algo>
__global__ void cukern_CFLtimestep(double *fluid, double *cs, double *out, int nx, int ntotal, int64_t slabpitch);

#define BLOCKDIM 8
#define GLOBAL_BLOCKDIM 128

__constant__ __device__ double geoParams[5];
#define GEO_DX geoParams[0]
#define GEO_DY geoParams[1]
#define GEO_DZ geoParams[2]
#define GEO_RIN geoParams[3]

// The process of acquiring page locked memory on the host is nauseatingly slow
// If this is set, grab these pointers once and never look again
#ifdef USE_STATIC_CHALLOC
static double *hostptrs[MAX_GPUS_USED];
// amount alloc'd in # doubles
static int hostptrLength[MAX_GPUS_USED];
#endif

#ifdef STANDALONE_MEX_FUNCTION
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	// Form of call: tau = cflTimestep(FluidManager, soundspeed gpu array, GeometryManager)

	MGArray fluid[5];
	GeometryParams geom;
	MGArray sndspeed;

	// At least 2 arguments expected
	// Input and result
	if((nlhs != 1) || (nrhs != 4))
		mexErrMsgTxt("Call must be tau = cflTimestep(FluidManager, soundspeed gpu array, GeometryManager, cfd_method);");

#ifdef USE_NVTX
		nvtxRangePush("Entering cflTimestep from mex");
#endif
	CHECK_CUDA_ERROR("entering cflTimestep");

	int worked;

	worked = MGA_accessFluidCanister(prhs[0], 0, &fluid[0]);
	if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) { mexErrMsgTxt("Dumping"); }
	worked = MGA_accessMatlabArrays(prhs, 1, 1, &sndspeed);
	if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) { mexErrMsgTxt("Dumping"); }

	geom = accessMatlabGeometryClass(prhs[2]);
	int meth = (int)*mxGetPr(prhs[3]);

	const mxArray *gdr = mxGetProperty(prhs[2], 0, "globalDomainRez");
	double *globrez = mxGetPr(gdr);

    mwSize outputDims[2];
    outputDims[0] = 1;
    outputDims[1] = 1;
    plhs[0] = mxCreateNumericArray (2, outputDims, mxDOUBLE_CLASS, mxREAL);

    double *timeStep = mxGetPr(plhs[0]);

	worked = computeLocalCFLTimestep(&fluid[0], &sndspeed, &geom, meth, globrez, timeStep);

}
#endif


int computeLocalCFLTimestep(MGArray *fluid, MGArray *csound, GeometryParams *geom, int method, double *globalResolution, double *tstep)
{
	int i;
	int sub[6];
	int worked;

	double geoarray[5];
	geoarray[0] = geom->h[0];
	geoarray[1] = geom->h[1];
	geoarray[2] = geom->h[2];
	geoarray[3] = geom->Rinner;

    dim3 blocksize, gridsize;
    blocksize.x = GLOBAL_BLOCKDIM; blocksize.y = blocksize.z = 1;

    // Launches enough blocks to fully occupy the GPU
    gridsize.x = 128;
    gridsize.y = gridsize.z =1;

    // Allocate enough pinned memory to hold results
    double *blkA[fluid->nGPUs];
    int hblockElements = gridsize.x;

    int spacedim = 0;
    if(globalResolution[1] > 1) spacedim = 1;
    if(globalResolution[2] > 1) spacedim = 2;

    int gt = 0;
    if(geom->shape == CYLINDRICAL) gt = 1;
    int ctype = spacedim + 3*(gt + 2*(method-1)); // value in 0..17

    int numBlocks[fluid->nGPUs];

    for(i = 0; i < fluid->nGPUs; i++) {
    	hipSetDevice(fluid->deviceID[i]);
    	worked = CHECK_CUDA_ERROR("hipSetDevice()");
    	if(worked != SUCCESSFUL) break;
    	// This DOES grow the allocation "dumbly" however it is extremely unlikely that reallocation will be necessary more than once, if that
#ifdef USE_STATIC_CHALLOC
    	if(hblockElements > hostptrLength[i]) {
    		if(hostptrLength[i] > 0) {
    			hipHostFree(hostptrs[i]);
    			CHECK_CUDA_ERROR("hipHostFree()");
    		}
    		hipHostMalloc((void **)&hostptrs[i], hblockElements * sizeof(double));
    		hostptrLength[i] = hblockElements;
    		worked = CHECK_CUDA_ERROR("CFL malloc doubles");
    		if(worked != SUCCESSFUL) break;
    	}
    	blkA[i] = hostptrs[i];
#else
    	hipHostMalloc((void **)&blkA[i], hblockElements * sizeof(double));
    	worked = CHECK_CUDA_ERROR("CFL malloc doubles");
    	if(worked != SUCCESSFUL) break;
#endif

    	hipMemcpyToSymbol(HIP_SYMBOL(geoParams), &geoarray[0], 5*sizeof(double), 0, hipMemcpyHostToDevice);
    	worked = CHECK_CUDA_ERROR("cfl const memcpy");
    	if(worked != SUCCESSFUL) break;
    	calcPartitionExtent(&fluid[0], i, &sub[0]);

    	gridsize.x = ROUNDUPTO(fluid[0].partNumel[i], blocksize.x) / blocksize.x;
    	if(gridsize.x > 128) gridsize.x = 128;

    	numBlocks[i] = gridsize.x;
    	double *fpi = fluid[0].devicePtr[i];
    	double *spi = csound->devicePtr[i];

    	switch(ctype) {
    	case 0:  cukern_CFLtimestep<1, SQUARE,      METHOD_HLL   ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 1:  cukern_CFLtimestep<2, SQUARE,      METHOD_HLL   ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 2:  cukern_CFLtimestep<3, SQUARE,      METHOD_HLL   ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 3:  cukern_CFLtimestep<1, CYLINDRICAL, METHOD_HLL   ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 4:  cukern_CFLtimestep<2, CYLINDRICAL, METHOD_HLL   ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 5:  cukern_CFLtimestep<3, CYLINDRICAL, METHOD_HLL   ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 6:  cukern_CFLtimestep<1, SQUARE,      METHOD_HLL   ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 7:  cukern_CFLtimestep<2, SQUARE,      METHOD_HLLC  ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 8:  cukern_CFLtimestep<3, SQUARE,      METHOD_HLLC  ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 9:  cukern_CFLtimestep<1, CYLINDRICAL, METHOD_HLLC  ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 10: cukern_CFLtimestep<2, CYLINDRICAL, METHOD_HLLC  ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 11: cukern_CFLtimestep<3, CYLINDRICAL, METHOD_HLLC  ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 12: cukern_CFLtimestep<1, SQUARE,      METHOD_HLLC  ><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 13: cukern_CFLtimestep<2, SQUARE,      METHOD_XINJIN><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 14: cukern_CFLtimestep<3, SQUARE,      METHOD_XINJIN><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 15: cukern_CFLtimestep<1, CYLINDRICAL, METHOD_XINJIN><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 16: cukern_CFLtimestep<2, CYLINDRICAL, METHOD_XINJIN><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	case 17: cukern_CFLtimestep<3, CYLINDRICAL, METHOD_XINJIN><<<gridsize, blocksize>>>(fpi, spi, blkA[i], sub[3], fluid[0].partNumel[i], fluid[0].slabPitch[i] / 8); break;
    	default:
    		PRINT_FAULT_HEADER;
    		printf("ctype ended up being %i, which is invalid because ctype must be int in [0,17].\n", ctype);
    		PRINT_FAULT_FOOTER;
    		worked = ERROR_INVALID_ARGS;
    		break;
    	}
    	worked = CHECK_CUDA_LAUNCH_ERROR(blocksize, gridsize, &fluid[0], i, "CFL max finder for Riemann solvers");
    	if(worked != SUCCESSFUL) break;
    }
    if(worked != SUCCESSFUL) { return worked; }

    double tmin = 1e38;

    int j;
    for(i = 0; i < fluid->nGPUs; i++) {
    	// They were all fired off at once, but we have to make sure each is done before
    	// reading the results of course
    	worked = hipSetDevice(fluid->deviceID[i]);
    	if(worked != SUCCESSFUL) { break; }
    	worked = hipDeviceSynchronize();
    	if(worked != SUCCESSFUL) { break; }

    	for(j = 0; j < numBlocks[i]; j++) {
    		tmin = (tmin < blkA[i][j]) ? tmin : blkA[i][j];
    	}
#ifdef USE_STATIC_CHALLOC

#else
    	hipHostFree(blkA[i]);
    	worked = CHECK_CUDA_ERROR("hipHostFree");
#endif
    }
    if(worked != SUCCESSFUL) { return worked; }

    double trueMin;
    MPI_Allreduce((void *)&tmin, (void *)&trueMin, 1, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);

    if((method == 1) || (method == 2)) { // HLL or HLLC
    	trueMin = trueMin / 2;
    	// Waves cannot be let to go more than HALF a cell: otherwise in principle two waves could collide,
    	// and emit a fast-moving signal that might propagate back to the fluxing interface.
    	// The method is, however, likely to be stable up to a twice this timestep. But likely is NOT good enough.
    }
#ifndef USE_SSPRK
    trueMin = trueMin / 2;
    // If we are using explicit midpoint, the timestep must be halved again to remain TVD
#endif
    tstep[0] = trueMin;

#ifdef SYNCMEX
    MGA_sledgehammerSequentialize(&fluid[0]);
#endif
#ifdef USE_NVTX
    nvtxRangePop();
#endif

    return worked;
}

template <int dimension>
__device__ __inline__ double getMagnitudeMomentum(double *base, int64_t pitch)
{
	double p, q;
	p = base[2*pitch]; // p_x or p_r
	if(dimension == 1) {
		return fabs(p);
	} else {

		p=p*p;
		q = base[3*pitch];
		p=p+q*q;

		if(dimension > 2) {
			q = base[4*pitch];
			p=p+q*q;
		}
		return sqrt(p);
	}
}

// 3 dims x 2 shapes x 3 algorithms = 18 kernels total
template <int simulationDimension, geometryType_t shape, FluidMethods algo>
__global__ void cukern_CFLtimestep(double *fluid, double *cs, double *out, int nx, int ntotal, int64_t slabpitch)
{
	unsigned int tix = threadIdx.x;
	int x = blockIdx.x * blockDim.x + tix; // address
	int blockhop = blockDim.x * gridDim.x;         // stepsize

	__shared__ double dtLimit[GLOBAL_BLOCKDIM];

	double u, v, w;
	double localTmin = 1e37;

	dtLimit[tix] = 1e37;

	if(x >= ntotal) return; // This is unlikely but we may get a stupid-small resolution

	fluid += x; // compute base offset
	cs += x;

	if((algo == METHOD_HLL) || (algo == METHOD_HLLC)) {
		if(shape == SQUARE) { // compute h once
			v = GEO_DX;
			if(simulationDimension > 1) { if(GEO_DY < v) v = GEO_DY; }
			if(simulationDimension > 2) { if(GEO_DZ < v) v = GEO_DZ; }
		}
		if(shape == CYLINDRICAL) { // Compute what we can compute just once
			v = GEO_DX;
			if(simulationDimension == 3) v = (v < GEO_DZ) ? v : GEO_DZ;
		}
	}

	while(x < ntotal) {
		if((algo == METHOD_HLL) || (algo == METHOD_HLLC)) {
			// get max signal speed
			u = getMagnitudeMomentum<simulationDimension>(fluid, slabpitch) / fluid[0] + cs[0]; // |v| + c

			// Identify local constraint on dt < dx / c_signal
			if(shape == SQUARE) {
				u = v / u;
			}
			if(shape == CYLINDRICAL) {
				w = (GEO_RIN + (x % nx) *GEO_DX)*GEO_DY; // r dtheta changes with r...
				w = (w < v) ? w : v;
				u = w / u;
			}
		}
		if(algo == METHOD_XINJIN) {
			double rho = fluid[0];
			// get max signal speed
			u = GEO_DX / ( fabs(fluid[2*slabpitch])/rho + cs[0] );

			if(simulationDimension > 1) {
				if(shape == SQUARE) {
					v = GEO_DY / ( fabs(fluid[3*slabpitch])/rho + cs[0] );
				}
				if(shape == CYLINDRICAL){
					v = (GEO_RIN + (x % nx)*GEO_DX)*GEO_DY / ( fabs(fluid[3*slabpitch])/rho + cs[0] );
				}
				u = (u < v) ? u : v;
			}
			if(simulationDimension > 2) {
				v = GEO_DZ / ( fabs(fluid[4*slabpitch])/rho + cs[0] );
				u = (u < v) ? u : v;
			}
		}

		// Each thread keeps running track of minimum dt
		localTmin = (u < localTmin) ? u : localTmin;

		fluid += blockhop;
		cs    += blockhop;
		x += blockhop; // skip the first block since we've already done it.
	}

	dtLimit[tix] = localTmin;

	__syncthreads();

	x = GLOBAL_BLOCKDIM / 2;
	while(x > 16) {
		if(tix >= x) return;
		__syncthreads();
		if(dtLimit[tix+x] < dtLimit[tix]) { dtLimit[tix] = dtLimit[tix+x]; }
		x=x/2;
	}

	if(tix >= 16) return;

	// We have one halfwarp (16 threads) remaining, proceed synchronously
	// cuda-memcheck --racecheck whines bitterly about this but because of warp synchronicity
	// there is no RAW problem.
	if(dtLimit[tix+16] < dtLimit[tix]) { dtLimit[tix] = dtLimit[tix+16]; } if(tix >= 8) return;
	//__syncthreads();
	if(dtLimit[tix+8] < dtLimit[tix])  { dtLimit[tix] = dtLimit[tix+8 ]; } if(tix >= 4) return;
	//__syncthreads();
	if(dtLimit[tix+4] < dtLimit[tix])  { dtLimit[tix] = dtLimit[tix+4 ]; } if(tix >= 2) return;
	//__syncthreads();
	if(dtLimit[tix+2] < dtLimit[tix])  { dtLimit[tix] = dtLimit[tix+2 ]; } if(tix) return;
	//__syncthreads();
	out[blockIdx.x] = (dtLimit[1] < dtLimit[0]) ? dtLimit[1] : dtLimit[0];

}
