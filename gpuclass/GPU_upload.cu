#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#ifdef UNIX
#include <stdint.h>
#include <unistd.h>
#endif
#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "cudaCommon.h"

// GPU_Tag = GPU_upload(host_array[double], device IDs[integers], [integer halo dim, integer partition direction])

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	// At least 2 arguments expected
	// Input and result
	if((nlhs != 1) || (nrhs < 1)) { mexErrMsgTxt("Form: result_tag = GPU_upload(host array [, device list [, (halo [,partition direct [, clone_partitions]])]])"); }

	CHECK_CUDA_ERROR("entering GPU_upload");

	MGArray m;

	// Default to no halo, X partition, add exterior halo
	m.haloSize = 0;
	m.partitionDir = PARTITION_X;
	m.addExteriorHalo = 1;
	m.vectorComponent = 0; // default, poke using GPU_Type.updateVectorComponent(n)
	int forceClone = 0;

	if(nrhs >= 3) {
		int a = mxGetNumberOfElements(prhs[2]);
		double *d = mxGetPr(prhs[2]);

		if(a >= 1) {
			m.haloSize = (int)*d;
			if(m.haloSize < 0) {
				printf("WARNING: Halo size %i is being clamped to zero.\n", m.haloSize);
				m.haloSize = 0;
			}

		}
		if(a >= 2) {
			m.partitionDir = (int)d[1];
			if((m.partitionDir < 1) || (m.partitionDir > 3)) m.partitionDir = PARTITION_X;
		}
		if(a >= 3) {
			// addExteriorHalo should be false iff #procs(partition direction) > 1
			m.addExteriorHalo = (int)d[2];
		}
		if(a >= 4) {
			forceClone = (int)d[3];
		}
	}

	// Default to circular boundary conditions
	m.mpiCircularBoundaryBits = 63;

	// With any new upload, assume this is the XYZ orientation
	m.permtag = 1;
	MGA_permtagToNums(m.permtag, &m.currentPermutation[0]);

	// Default to entire array on current device
	m.nGPUs = 1;
	hipGetDevice(&m.deviceID[0]);
	// But of course we may partition it otherwise
	if(nrhs >= 2) {
		int j;
		double *g = mxGetPr(prhs[1]);
		m.nGPUs = mxGetNumberOfElements(prhs[1]);
		for(j = 0; j < m.nGPUs; j++) {
			m.deviceID[j] = (int)g[j];
			m.devicePtr[j] = 0x0;
		}
	}

	double *hmem = mxGetPr(prhs[0]);
	int nd = mxGetNumberOfDimensions(prhs[0]);
	if(nd > 3) mexErrMsgTxt("Array dimensionality > 3 unsupported.");
	const mwSize *idims = mxGetDimensions(prhs[0]);
	int i;
	for(i = 0; i < nd; i++) { m.dim[i] = idims[i]; }
	for(;      i < 3; i++) { m.dim[i] = 1; }

	// If we are already cloning, multiply the size in the partition direct by #GPUs
	if(forceClone) {
		m.dim[m.partitionDir-1] = m.dim[m.partitionDir-1] * m.nGPUs;
	}

	// If the size in the partition direction is 1, clone it instead
	if((m.dim[m.partitionDir-1] == 1) && (m.nGPUs > 1)) {
		m.haloSize = 0;
		m.dim[m.partitionDir-1] = m.nGPUs;
		forceClone = 1;
	}

	m.numel = m.dim[0]*m.dim[1]*m.dim[2];
	int sub[6];
	for(i = 0; i < m.nGPUs; i++) {
		calcPartitionExtent(&m, i, &sub[0]);
		m.partNumel[i] = sub[3]*sub[4]*sub[5];
	}
	m.numSlabs = 1;

	MGArray *dest = MGA_createReturnedArrays(plhs, 1, &m);

	int worked;
	if(forceClone) {
		worked = MGA_uploadArrayToGPU(hmem, dest, 0);
	} else {
		worked = MGA_uploadArrayToGPU(hmem, dest, -1);
	}
	if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) {
		mexErrMsgTxt("Attempt to upload Matlab array to GPU was unsuccessful.\n");
		return;
	}
	if(forceClone) {
	    worked = MGA_distributeArrayClones(dest, 0);
	    if(CHECK_IMOGEN_ERROR(worked) != SUCCESSFUL) {
	    	mexErrMsgTxt("Redistribution of cloned array failed!\n");
	    }
	}


	return;
}
